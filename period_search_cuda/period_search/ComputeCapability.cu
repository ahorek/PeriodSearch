#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include "ComputeCapability.h"

Cc::Cc(const hipDeviceProp_t& deviceProp)
{
	this->cudaVersion = CUDART_VERSION;
	deviceCcMajor = deviceProp.major;
	deviceCcMinor = deviceProp.minor;
}

#if defined (_MSC_VER) & (_MSC_VER < 1900) // Visual Studio 2012 or previous
	Cc::~Cc(){}
#else
	Cc::~Cc() 
	{ 
		// Your cleanup code here, if any. 
	}
#endif


int Cc::GetSmxBlock() const
{
	auto result = 0;
	if (cudaVersion >= 12000 && cudaVersion < 13000)
	{
		result = GetSmxBlockCuda12();
	}
	else if (cudaVersion >= 11000 && cudaVersion < 12000)
	{
		result = GetSmxBlockCuda11();
	}
	else if (cudaVersion >= 10000 && cudaVersion < 11000)
	{
		result = GetSmxBlockCuda10();
	}
	else if (cudaVersion >= 6000 && cudaVersion < 10000)
	{
		result = GetSmxBlockCuda6();
	}

	return result;
}


int Cc::GetSmxBlockCuda12() const
{
	auto smxBlock = 0;
	switch (deviceCcMajor)
	{
	case 10:
		smxBlock = 16; // Fall back to safe value
		break;
	case 9:
		smxBlock = GetSmxBlockCc9(); // Hopper
		break;
	case 8:
		smxBlock = GetSmxBlockCc8(); // Ampere micro architecture CC 8.0, 8.6; Ada Lovelace - CC 8.9
		break;
	case 7:
		smxBlock = GetSmxBlockCc7(); // 7.0, 7.2: Volta; 7.5: Turing 
		break;
	case 6:
		smxBlock = GetSmxBlockCc6(); // Pascal
		break;
	case 5:
		smxBlock = GetSmxBlockCc5(); // Maxwell
		break;
	default:
		Exit();
		break;
	}

	return smxBlock;
}

int Cc::GetSmxBlockCuda11() const
{
	auto smxBlock = 0;
	switch (deviceCcMajor)
	{
	case 10:
		smxBlock = 16; // Fall back to safe value
		break;
	case 9:
		smxBlock = GetSmxBlockCc9(); // Hopper
		break;
	case 8:
		smxBlock = GetSmxBlockCc8(); // Ampere micro architecture CC 8.0, 8.6; Ada Lovelace - CC 8.9
		break;
	case 7:
		smxBlock = GetSmxBlockCc7(); // 7.0, 7.2: Volta; 7.5: Turing 
		break;
	case 6:
		smxBlock = GetSmxBlockCc6(); // Pascal
		break;
	case 5:
		smxBlock = GetSmxBlockCc5(); // Maxwell
		break;
	default:
		Exit();
		break;
	}

	return smxBlock;
}



int Cc::GetSmxBlockCuda10() const
{
	auto smxBlock = 0;
	switch (deviceCcMajor)
	{
	case 10:
	case 9:
		smxBlock = 16; // Fall back to safe value
		break;
	case 8:
		smxBlock = GetSmxBlockCc8();
		break;
	case 7:
		smxBlock = GetSmxBlockCc7();
		break;
	case 6:
		smxBlock = GetSmxBlockCc6();
		break;
	case 5:
		smxBlock = GetSmxBlockCc5();
		break;
	case 3:
		smxBlock = GetSmxBlockCc3(); // Kepler
		break;
	default:
		Exit();
		break;
	}

	return smxBlock;
}

int Cc::GetSmxBlockCuda6() const
{
	auto smxBlock = 0;
	switch (deviceCcMajor)
	{
	case 10:
	case 9:
	case 8:
	case 7:
	case 6:
	case 5:
		smxBlock = 16; // Fall back to safe value 
		break;
	case 3:
		smxBlock = GetSmxBlockCc3(); // Kepler
		break;
	case 2:
		smxBlock = GetSmxBlockCc2(); // Fermi
		break;
	case 1:
		smxBlock = GetSmxBlockCc1(); // Tesla
		break;
	default:
		Exit();
		break;
	}

	return smxBlock;
}

int Cc::GetSmxBlockCc9() const
{
	auto smxBlock = 0;
	switch (deviceCcMinor)
	{
	case 0:
		smxBlock = 32;	// Hopper
		break;
	default:
		Exit();
		break;
	}

	return smxBlock;
}

int Cc::GetSmxBlockCc8() const
{
	auto smxBlock = 0;
	switch (deviceCcMinor)
	{
	case 0:
		smxBlock = 32;	// Tesla A100 | occupancy 100% = 32 blocks per SMX
		break;
	case 6:
	case 7:
		smxBlock = 16;	// GeForce RTX 3080 etc.; Quadro A6000 | occupancy 100% = 16 blocks per SMX
		break;
	case 8:
		smxBlock = 16;	// ZLuda
		break;
	case 9:
		smxBlock = 24;	// GeForce RTX 4090, RTX 4080 16GB; RTX 6000 Ada | occupancy 100% = 24 blocks per SMX
		break;
	default:
		Exit();
		break;
	}

	return smxBlock;
}

int Cc::GetSmxBlockCc7() const
{
	auto smxBlock = 0;
	switch (deviceCcMinor)
	{
		case 0:				// CC 7.0 & 7.2, occupancy 100% = 32 blocks per SMX
		case 2:
			smxBlock = 32;
			break;
		case 5:				// CC 7.5, occupancy 100% = 16 blocks per SMX
			smxBlock = 16;
			break;
		default:			
			Exit();
			break;
	}

	return smxBlock;
}

int Cc::GetSmxBlockCc6() const
{
	auto smxBlock = 0;
	switch (deviceCcMinor)
	{
		case 0:
		case 1:
		case 2:
			smxBlock = 32; //occupancy 100% = 32 blocks per SMX
			break;
		default:
			Exit();
			break;
	}

	return smxBlock;
}

int Cc::GetSmxBlockCc5() const
{
	auto smxBlock = 0;
	switch (deviceCcMinor)
	{
	// TODO: There is something rot in Denmark...
//#if (CUDART_VERSION < 11000)
		case 0:
		case 2:
//#endif
		case 3:
			smxBlock = 32; //occupancy 100% = 32 blocks per SMX, instead as previous was 16 blocks per SMX which led to only 50%
			break;
		default:
			Exit();
			break;
	}

	return smxBlock;
}

int Cc::GetSmxBlockCc3() const
{
	auto smxBlock = 0;
	switch(deviceCcMinor)
	{
		//CC 3.0, 3.2, 3.5 & 3.7
		case 0:
		case 2:
		case 3:
		case 5:
		case 7:
			smxBlock = 16; //occupancy 100% = 16 blocks per SMX
			break;
		default:
			Exit();
			break;			
	}

	return smxBlock;
}

int Cc::GetSmxBlockCc2() const
{
	auto smxBlock = 0;
	switch(deviceCcMinor)
	{
		//CC 2.0, 2.1
		case 0:
		case 1:
			smxBlock = 8; //occupancy 100% = 8 blocks per SMX
			break;
		default:
			Exit();
			break;
	}

	return smxBlock;
}

int Cc::GetSmxBlockCc1() const
{
	auto smxBlock = 0;
	switch(deviceCcMinor)
	{
		//CC 1.0, 1.1, 1.2, 1.3
		case 0:
		case 1:
		case 2:
		case 3:
			smxBlock = 8; //occupancy 100% = 8 blocks per SMX
			break;
		default:
			Exit();
			break;
	}

	return smxBlock;
}

void Cc::Exit() const
{
	fprintf(stderr, "Unsupported Compute Capability (CC) detected (%d.%d).\n", deviceCcMajor, deviceCcMinor);
	exit(1);
}
