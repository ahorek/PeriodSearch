#include "hip/hip_runtime.h"
/* N.B. The foll. L-M routines are modified versions of Press et al.
   converted from Mikko's fortran code

   8.11.2006
*/

#include <hip/hip_runtime.h>
#include "globals_CUDA.h"
#include "declarations_CUDA.h"
#include <>
#include <stdio.h>

__device__ int __forceinline__ mrqmin_1_end(freq_context *__restrict__ CUDA_LCC, int ma, int mfit, int mfit1, const int block)
{
	int bid = blockIdx();

	if (__ldg(&isAlamda[bid]))
	{
		int n = threadIdx.x + 1;
		double *__restrict__ ap = atry[bid] + n;
		double const *__restrict__ cgp = CUDA_LCC->cg + n;
#pragma unroll 1
		while (n <= ma - block)
		{
			ap[0] = cgp[0];
			ap[block] = cgp[block];
			n += 2 * block;
			ap += 2 * block;
			cgp += 2 * block;
		}
		if (n <= ma)
		{
			ap[0] = cgp[0];
		}
	}

	double ccc = 1 + __ldg(&Alamda[bid]);

	int ixx = mfit1 + threadIdx.x + 1;

	double *__restrict__ a = CUDA_LCC->covar + ixx;
	double const *__restrict__ b = CUDA_LCC->alpha + ixx;
#pragma unroll 2
	while (ixx < mfit1 * mfit1 - (UNRL - 1) * block)
	{
		int i;
		double t[UNRL];
		for (i = 0; i < UNRL; i++)
		{
			t[i] = b[0];
			b += block;
		}
		for (i = 0; i < UNRL; i++)
		{
			if ((ixx + i * block) % (mfit1 + 1) == 0)
				a[0] = ccc * t[i];
			else
				a[0] = t[i];
			a += block;
		}
		ixx += UNRL * block;
	}
#pragma unroll 3
	while (ixx < mfit1 * mfit1)
	{
		double t = b[0];
		if (ixx % (mfit1 + 1) == 0)
			*a = ccc * t;
		else
			*a = t;

		a += block;
		b += block;
		ixx += block;
	}

	int xx = threadIdx.x + 1;
	double const *__restrict__ bp;
	double *__restrict__ dap;
	bp = CUDA_LCC->beta + xx;
	dap = CUDA_LCC->da + xx;
#pragma unroll 1
	while (xx <= mfit - block)
	{
		dap[0] = bp[0];
		dap[block] = bp[block];
		bp += 2 * block;
		dap += 2 * block;
		xx += 2 * block;
	}
	if (xx <= mfit)
	{
		*dap = bp[0];
		bp += block;
		dap += block;
		xx += block;
	}

	__syncwarp();

	int err_code = gauss_errc(CUDA_LCC, ma);
	if (err_code)
	{
		return err_code;
	}

	int n = threadIdx.x + 1;
	int const *__restrict__ iap = CUDA_ia + n;
	double *__restrict__ ap = atry[bid] + n;
	double const *__restrict__ cgp = CUDA_LCC->cg + n;
	double const *__restrict__ ddap = CUDA_LCC->da + n - 1;
#pragma unroll 1
	while (n <= ma - block)
	{
		if (*iap)
			*ap = cgp[0] + ddap[0];
		if (iap[block])
			ap[block] = cgp[block] + ddap[block];
		n += 2 * block;
		iap += 2 * block;
		ap += 2 * block;
		cgp += 2 * block;
		ddap += 2 * block;
	}
	// #pragma unroll 2
	if (n <= ma)
	{
		if (*iap)
			*ap = cgp[0] + ddap[0];
	}
	//__syncthreads();

	return err_code;
}

// clean pointers and []'s
// threadify loops
__device__ void __forceinline__ mrqmin_2_end(freq_context *__restrict__ CUDA_LCC, int ma, int bid)
{
	int j, k, l; //, bid = blockIdx();
	int mf = CUDA_mfit, mf1 = CUDA_mfit1;

	if (Chisq[bid] < Ochisq[bid])
	{
		double rai = CUDA_Alamda_incr;
		double const *__restrict__ dap = CUDA_LCC->da + 1 + threadIdx.x;
		double *__restrict__ dbp = CUDA_LCC->beta + 1 + threadIdx.x;
#pragma unroll 1
		for (j = threadIdx.x; j < mf - CUDA_BLOCK_DIM; j += CUDA_BLOCK_DIM)
		{
			double v1 = dap[0];
			double v2 = dap[CUDA_BLOCK_DIM];
			dbp[0] = v1;
			dbp[CUDA_BLOCK_DIM] = v2;
			dbp += 2 * CUDA_BLOCK_DIM;
			dap += 2 * CUDA_BLOCK_DIM;
		}
		if (j < mf)
			*dbp = dap[0];

		rai = __drcp_rn(rai); /// 1.0/rai;

		double const *__restrict__ cvp = CUDA_LCC->covar + mf1 + threadIdx.x;

		double *__restrict__ ap = CUDA_LCC->alpha + mf1 + threadIdx.x;

		double const *__restrict__ cvpo = cvp + 1;

		double *apo = ap + 1;

		Alamda[bid] = __ldg(&Alamda[bid]) * rai;

#pragma unroll 1
		for (j = 0; j < mf; j++)
		{
			cvp = cvpo;
			ap = apo;
#pragma unroll 1
			for (k = threadIdx.x; k < mf - CUDA_BLOCK_DIM; k += CUDA_BLOCK_DIM)
			{
				double v1 = cvp[0];
				double v2 = cvp[CUDA_BLOCK_DIM];
				ap[0] = v1;
				ap[CUDA_BLOCK_DIM] = v2;
				cvp += 2 * CUDA_BLOCK_DIM;
				ap += 2 * CUDA_BLOCK_DIM;
			}

			if (k < mf)
				__stwb(ap, __ldca(cvp)); //[0]; //ldcs

			cvpo += mf + 1;
			apo += mf + 1;
		}

		double const *__restrict__ atp = atry[bid] + 1 + threadIdx.x;

		double *__restrict__ cgp = CUDA_LCC->cg + 1 + threadIdx.x;

#pragma unroll 1
		for (l = threadIdx.x; l < ma - CUDA_BLOCK_DIM; l += CUDA_BLOCK_DIM)
		{
			double v1 = atp[0];
			double v2 = atp[CUDA_BLOCK_DIM];
			cgp[0] = v1;
			cgp[CUDA_BLOCK_DIM] = v2;
			atp += CUDA_BLOCK_DIM;
			cgp += CUDA_BLOCK_DIM;
		}

		if (l < ma)
			*cgp = atp[0];
	}
	else if (threadIdx.x == 0)
	{
		double a, c;
		a = CUDA_Alamda_incr * __ldg(&Alamda[bid]);
		c = Ochisq[bid];
		Alamda[bid] = a;
		Chisq[bid] = c;
	}

	return;
}