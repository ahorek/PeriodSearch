#include "hip/hip_runtime.h"
/* Convexity regularization function

   8.11.2006
*/

#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include "globals_CUDA.h"
#include "declarations_CUDA.h"
#include "cuda_vars.cuh"
#include <>


__device__ double __forceinline__ conv(freq_context *__restrict__ CUDA_LCC, int nc, double *__restrict__ dyda, int bid)
{
  int i, j;
  //__shared__ double res[CUDA_BLOCK_DIM];
  double tmp, tmp2; //, dtmp, dtmp2;
  int nf = CUDA_Numfac, nf1 = CUDA_Numfac1, nco = CUDA_Ncoef;

  j = bid * nf1 + threadIdx.x + 1;
  int xx = threadIdx.x + 1;
  tmp = 0, tmp2 = 0;
  // double * __restrict__ areap = CUDA_Area + j;
  double *__restrict__ areap = &(Areag[bid][threadIdx.x + 1]);
  double *__restrict__ norp = CUDA_Nor[nc] + xx;
#pragma unroll 4
  while (xx <= nf - CUDA_BLOCK_DIM)
  {
    double a0, a1, n0, n1;
    a0 = areap[0];
    n0 = norp[0];
    a1 = areap[CUDA_BLOCK_DIM];
    n1 = norp[CUDA_BLOCK_DIM];
    tmp += a0 * n0;  // areap[0] * norp[0];
    tmp2 += a1 * n1; // areap[CUDA_BLOCK_DIM] * norp[CUDA_BLOCK_DIM];
    xx += 2 * CUDA_BLOCK_DIM;
    areap += 2 * CUDA_BLOCK_DIM;
    norp += 2 * CUDA_BLOCK_DIM;
  }
  // #pragma unroll 1
  if (xx <= nf)
  {
    tmp += areap[0] * norp[0]; // CUDA_Area[j] * CUDA_Nor[nc][xx];
  }

  tmp += tmp2;

  tmp += __shfl_down_sync(0xffffffff, tmp, 16);
  tmp += __shfl_down_sync(0xffffffff, tmp, 8);
  tmp += __shfl_down_sync(0xffffffff, tmp, 4);
  tmp += __shfl_down_sync(0xffffffff, tmp, 2);
  tmp += __shfl_down_sync(0xffffffff, tmp, 1);
  /*
#if CUDA_BLOCK_DIM == 128
  __shared__ double mm, nn, vv;
  if(threadIdx.x == 96)
    vv = tmp;
  if(threadIdx.x == 64)
    nn = tmp;
  if(threadIdx.x == 32)
    mm = tmp;
  __syncthreads();
  if(threadIdx.x == 0)
    tmp += mm + nn + vv;
#endif
#if CUDA_BLOCK_DIM == 64
  __shared__ double nn;
  if(threadIdx.x == 32)
    nn = tmp;
  __syncthreads();
  if(threadIdx.x == 0)
    tmp += nn;
#endif
  */
  int ma = CUDA_ma, dg_block = CUDA_Dg_block;
  double *__restrict__ dg = CUDA_Dg, *__restrict__ darea = CUDA_Darea, *__restrict__ nor = CUDA_Nor[nc];
#pragma unroll 1
  for (j = 1; j <= ma; j++)
  {
    int m = blockIdx() * dg_block + j * nf1;
    double dtmp = 0, dtmp2 = 0;
    if (j <= nco)
    {
      int mm = m + threadIdx.x + 1;

      i = threadIdx.x + 1;
      double *__restrict__ dgp = dg + mm;
      double *__restrict__ dareap = darea + i;
      double *__restrict__ norp = nor + i;

#pragma unroll 4
      while (i <= nf - CUDA_BLOCK_DIM)
      {
        double g0, g1, a0, a1, n0, n1;
        g0 = dgp[0];
        a0 = dareap[0];
        n0 = norp[0];
        g1 = dgp[CUDA_BLOCK_DIM];
        a1 = dareap[CUDA_BLOCK_DIM];
        n1 = norp[CUDA_BLOCK_DIM];
        dtmp += (g0 * a0) * n0;
        dtmp2 += (g1 * a1) * n1;
        i += 2 * CUDA_BLOCK_DIM;
        dgp += 2 * CUDA_BLOCK_DIM;
        dareap += 2 * CUDA_BLOCK_DIM;
        ;
        norp += 2 * CUDA_BLOCK_DIM;
      }
      // #pragma unroll 1
      if (i <= nf) //; i += CUDA_BLOCK_DIM, mm += CUDA_BLOCK_DIM)
      {
        dtmp += dgp[0] * dareap[0] * norp[0]; // CUDA_Dg[mm] * CUDA_Darea[i] * CUDA_Nor[nc][i];
      }

      dtmp += dtmp2;

      dtmp += __shfl_down_sync(0xffffffff, dtmp, 16);
      dtmp += __shfl_down_sync(0xffffffff, dtmp, 8);
      dtmp += __shfl_down_sync(0xffffffff, dtmp, 4);
      dtmp += __shfl_down_sync(0xffffffff, dtmp, 2);
      dtmp += __shfl_down_sync(0xffffffff, dtmp, 1);
    }

    if (threadIdx.x == 0)
      dyda[j - 1] = dtmp;
  }

  return (tmp);
}
