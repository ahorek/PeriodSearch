#include "hip/hip_runtime.h"
/* Curvature function (and hence facet area) from Laplace series

   8.11.2006
*/

#include <hip/hip_runtime.h>
#include <math.h>
#include "globals_CUDA.h"


__device__ void __forceinline__ curv(freq_context const * __restrict__ CUDA_LCC, double * __restrict__ cg, int bid)
{
  int i, m, n, l, k;
  double g;
  
  int numfac = CUDA_Numfac, nf1 = CUDA_Numfac1, mm = CUDA_Mmax, lm = CUDA_Lmax;
  i = threadIdx.x + 1;
  double * __restrict__ CUDA_Fcp = CUDA_Fc[0] + i;
  double * __restrict__ CUDA_Fsp = CUDA_Fs[0] + i;
  double * __restrict__ CUDA_Dareap = CUDA_Darea + i;

#pragma unroll 1
  while(i <= numfac)
    {
      g = 0;
      n = 0;
      double const * __restrict__ cgp = cg + 1;
      double const * __restrict__ fcp = CUDA_Fcp;
      double const * __restrict__ fsp = CUDA_Fsp;

#pragma unroll 2
      for(m = 0; m <= mm; m++)
	{ 
	  double fcim = __ldca(&fcp[0]); //* //[m*(MAX_N_FAC + 1)]; //CUDA_Fc[m][i];
	  double fsim = __ldca(&fsp[0]); //[m*(MAX_N_FAC + 1)]; //CUDA_Fs[m][i];
	  double * __restrict__ CUDA_Plegp = &CUDA_Pleg[m][m][i]; //[MAX_LM + 1][MAX_LM + 1][MAX_N_FAC + 1];
#pragma unroll 3
	  for(l = m; l <= lm; l++)
	    {
	      n++;
	      double fsum = __ldca(cgp++) * fcim; //CUDA_Fc[i][m];
	      if(m > 0)
		{
		  n++;
		  fsum += __ldca(cgp++) * fsim; //CUDA_Fs[i][m];
		}
	      g += CUDA_Plegp[0] * fsum; //[m][l][i] * fsum; //CUDA_Pleg[m][l][i] * fsum;
	      CUDA_Plegp += (MAX_N_FAC + 1);
	    }
	  fcp += MAX_N_FAC + 1;
	  fsp += MAX_N_FAC + 1;
	}
      double dd = CUDA_Dareap[0];
      g = exp(g);
      dd *= g;
      double * __restrict__ dgp = CUDA_LCC->Dg + (nf1 + i);
      double const * __restrict__ dsphp = CUDA_Dsph[0] + i + MAX_N_FAC + 1;
      
      Areag[bid][i] = dd;
      k = 1;
#pragma unroll 1
      while(k <= n - (UNRL - 1))
	{
	  double a[UNRL];

#pragma unroll 
	  for(int nn = 0; nn < UNRL; nn++)
	    {
	      a[nn] = __ldca(dsphp) * g;
	      dsphp += (MAX_N_FAC + 1);
	    }
#pragma unroll 
	  for(int nn = 0; nn < UNRL; nn++)
	    {
	      __stwb(dgp, a[nn]);
	      dgp += nf1;
	    }
	  k += UNRL;
	}
#pragma unroll 3
      while(k <= n)
	{
	  __stwb(dgp, dsphp[0] * g);
	  dsphp += (MAX_N_FAC + 1);
	  k++;
	  dgp += nf1;
	}

      i += CUDA_BLOCK_DIM;
      CUDA_Fcp += CUDA_BLOCK_DIM;
      CUDA_Fsp += CUDA_BLOCK_DIM;
      CUDA_Dareap += CUDA_BLOCK_DIM;
    }
  //__syncwarp();
}