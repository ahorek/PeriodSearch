#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <math.h>

#include "constants.h"
#include "globals_CUDA.h"
#include "declarations_CUDA.h"
#include "cuda_vars.cuh"
#include <cstdio>

// vars

__device__ double Dblm[2][3][3][N_BLOCKS]; // OK, set by [tid], read by [bid]
__device__ double Blmat[3][3][N_BLOCKS];   // OK, set by [tid], read by [bid]

__device__ double CUDA_scale[N_BLOCKS][POINTS_MAX + 1];   // OK [bid][tid]
__device__ double ge[2][3][N_BLOCKS][POINTS_MAX + 1];     // OK [bid][tid]
__device__ double gde[2][3][3][N_BLOCKS][POINTS_MAX + 1]; // OK [bid][tid]
__device__ double jp_dphp[3][N_BLOCKS][POINTS_MAX + 1];   // OK [bid][tid]

__device__ double dave[N_BLOCKS][MAX_N_PAR + 1];
__device__ double atry[N_BLOCKS][MAX_N_PAR + 1];

__device__ double chck[N_BLOCKS];
__device__ int    isInvalid[N_BLOCKS];
__device__ int    isNiter[N_BLOCKS];
__device__ int    isAlamda[N_BLOCKS];
__device__ double Alamda[N_BLOCKS];
__device__ int    Niter[N_BLOCKS];
__device__ double iter_diffg[N_BLOCKS];
__device__ double rchisqg[N_BLOCKS]; // not needed
__device__ double dev_oldg[N_BLOCKS];
__device__ double dev_newg[N_BLOCKS];

__device__ double trial_chisqg[N_BLOCKS];
__device__ double aveg[N_BLOCKS];
__device__ int    npg[N_BLOCKS];
__device__ int    npg1[N_BLOCKS];
__device__ int    npg2[N_BLOCKS];

__device__ double Ochisq[N_BLOCKS];
__device__ double Chisq[N_BLOCKS];
__device__ double Areag[N_BLOCKS][MAX_N_FAC + 1];

//LFR
__managed__ int isReported[N_BLOCKS];
__managed__ double dark_best[N_BLOCKS];
__managed__ double per_best[N_BLOCKS];
__managed__ double dev_best[N_BLOCKS];
__managed__ double la_best[N_BLOCKS];
__managed__ double be_best[N_BLOCKS];


#ifdef NEWDYTEMP
__device__ double dytemp[POINTS_MAX + 1][40][N_BLOCKS];
#endif

#define CUDA_Nphpar 3

//global to all freq
__constant__ int CUDA_Ncoef, CUDA_Numfac, CUDA_Numfac1, CUDA_Dg_block;
__constant__ int CUDA_ma, CUDA_mfit, CUDA_mfit1, CUDA_lastone, CUDA_lastma, CUDA_ncoef0;
__constant__ double CUDA_cg_first[MAX_N_PAR + 1];
__constant__ int CUDA_n_iter_max, CUDA_n_iter_min, CUDA_ndata;
__constant__ double CUDA_iter_diff_max;
__constant__ double CUDA_conw_r;
__constant__ int CUDA_Lmax, CUDA_Mmax;
__constant__ double CUDA_lcl, CUDA_Alamda_start, CUDA_Alamda_incr;  //, CUDA_Alamda_incrr;
__constant__ double CUDA_Phi_0;
__constant__ double CUDA_beta_pole[N_POLES + 1];
__constant__ double CUDA_lambda_pole[N_POLES + 1];

__device__ double CUDA_par[4];
__device__ int CUDA_ia[MAX_N_PAR + 1];
__device__ double CUDA_Nor[3][MAX_N_FAC + 1];
__device__ double CUDA_Fc[MAX_LM+1][MAX_N_FAC + 1];
__device__ double CUDA_Fs[MAX_LM+1][MAX_N_FAC + 1];
__device__ double CUDA_Pleg[MAX_LM + 1][MAX_LM + 1][MAX_N_FAC + 1];
__device__ double CUDA_Darea[MAX_N_FAC + 1];
__device__ double CUDA_Dsph[MAX_N_PAR + 1][MAX_N_FAC + 1];
__device__ double CUDA_ee[3][MAX_N_OBS + 1]; //[3][MAX_N_OBS+1];
__device__ double CUDA_ee0[3][MAX_N_OBS+1];
__device__ double CUDA_tim[MAX_N_OBS + 1];
__device__ double *CUDA_brightness/*[MAX_N_OBS+1]*/;
__device__ double *CUDA_sig/*[MAX_N_OBS+1]*/;
__device__ double *CUDA_Weight/*[MAX_N_OBS+1]*/;
//__device__ double *CUDA_Area;
__device__ double *CUDA_Dg;
__device__ int CUDA_End;
__device__ int CUDA_Is_Precalc;

//global to one thread
__device__ freq_context *CUDA_CC;

__global__ void CudaCalculatePrepare(int n_start, int n_max)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int n = n_start + tid;

  if(n > n_max)
    {
      isInvalid[tid] = 1;
      return;
    }
  else
    {
      isInvalid[tid] = 0;
    }

  per_best[tid] = 0; 
  dark_best[tid] = 0;
  la_best[tid] = 0;
  be_best[tid] = 0;
  dev_best[tid] = 1e40;
}

__global__
__launch_bounds__(1024)
  void CudaCalculatePreparePole(int m, double freq_start, double freq_step, int n_start)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  n_start += tid;
  auto CUDA_LCC = &CUDA_CC[tid];
  //auto CUDA_LFR = &CUDA_FR[tid];

  if(__ldg(&isInvalid[tid]))  
    {
      atomicAdd(&CUDA_End, 1);
      isReported[tid] = 0; //signal not to read result

      return;
    }

  //double period = ___drcp_rn(__ldg(&CUDA_freq[tid]));
  double period = ___drcp_rn(freq_start - (n_start - 1) * freq_step);
  double * __restrict__ cgp = CUDA_LCC->cg + 1;
  double const * __restrict__ cfp = CUDA_cg_first + 1;
  /* starts from the initial ellipsoid */
  int i;
  int ncoef = CUDA_Ncoef;
#pragma unroll 4
  for(i = 1; i <= ncoef - (UNRL - 1); i += UNRL)
    {
      double d[UNRL];
      int ii;
      for(ii = 0; ii < UNRL; ii++)
	d[ii] = *cfp++;
      for(ii = 0; ii < UNRL; ii++)
	*cgp++ = d[ii];
    }
#pragma unroll 3
  for( ; i <= ncoef; i++)
    {
      *cgp++ = *cfp++; //CUDA_cg_first[i];
    }

  
  /* The formulae use beta measured from the pole */
  /* conversion of lambda, beta to radians */
  *cgp++ = DEG2RAD * 90 - DEG2RAD * CUDA_beta_pole[m];
  *cgp++ = DEG2RAD * CUDA_lambda_pole[m];
   
  /* Use omega instead of period */
  *cgp++ = (24.0 * 2.0 * PI) / period;

#pragma unroll
  for(i = 1; i <= CUDA_Nphpar; i++)
    {
      *cgp++ = CUDA_par[i];
    }
  
  /* Use logarithmic formulation for Lambert to keep it positive */
  *cgp++ = CUDA_lcl; //log(CUDA_cl); 
  /* Lommel-Seeliger part */
  *cgp++ = 1;

  /* Levenberg-Marquardt loop */
  // moved to global iter_max,iter_min,iter_dif_max
  //
  rchisqg[tid] = -1;
  Alamda[tid] = -1;
  Niter[tid] = 0;
  iter_diffg[tid] = 1e40;
  dev_oldg[tid] = 1e30;
  dev_newg[tid] = 0;
  isReported[tid] = 0;
}

__global__ void CudaCalculateIter1Begin(int n_max)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(tid > n_max) return;

  if(__ldg(&isInvalid[tid])) 
    {
      return;
    }

  int niter = __ldg(&Niter[tid]);
  bool b_isniter = ((niter < CUDA_n_iter_max) && (iter_diffg[tid] > CUDA_iter_diff_max)) || (niter < CUDA_n_iter_min);
  isNiter[tid] = b_isniter;

  if(b_isniter)
    {
      if(__ldg(&Alamda[tid]) < 0)
	{
	  isAlamda[tid] = 1;
	  Alamda[tid] = CUDA_Alamda_start; /* initial alambda */
	}
      else
	isAlamda[tid] = 0;
    }
  else
    {
      if(!(__ldg(&isReported[tid])))
	{
	  atomicAdd(&CUDA_End, 1);
#ifdef _DEBUG
	  /*const int is_precalc = CUDA_Is_Precalc;
	    if(is_precalc)
	    {
	    printf("%d ", CUDA_End);
	    }*/
#endif
	  isReported[tid] = 1;
	}
    }
}

__global__
__launch_bounds__(768)
void CudaCalculateIter1Mrqmin1End(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return; //CUDA_LCC->isInvalid) return;

  if(!__ldg(&isNiter[bid])) return;

  /*gauss_err=*/
  mrqmin_1_end(CUDA_LCC, CUDA_ma, CUDA_mfit, CUDA_mfit1, CUDA_BLOCK_DIM);
}

__global__ void CudaCalculateIter1Mrqmin2End(void)
{
  //int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];
  if(__ldg(&isInvalid[bid])) return;

  if(!__ldg(&isNiter[bid])) return;

  mrqmin_2_end(CUDA_LCC, CUDA_ma, bid);

  __syncwarp();
  if(threadIdx.x == 0)
    Niter[bid]++;
  //CUDA_LCC->Niter++;
}

__global__
__launch_bounds__(512)
void CudaCalculateIter1Mrqcof1Start(void)
{
  int tid = blockIdx() * blockDim.x + threadIdx.x;

  if(tid < blockDim.y * gridDim.x)
    {
      auto CUDA_LCC = &CUDA_CC[tid];
 
      double *a = CUDA_LCC->cg;
      blmatrix(a[CUDA_ma-4-CUDA_Nphpar], a[CUDA_ma-3-CUDA_Nphpar], tid);
    }

  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];
  
  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  if(threadIdx.x == 0)
    {
      trial_chisqg[bid] = 0;
      npg[bid] = 0;
      npg1[bid] = 0;
      npg2[bid] = 0;
      aveg[bid] = 0;
    }

  mrqcof_start(CUDA_LCC, CUDA_LCC->cg, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}

__global__ void CudaCalculateIter1Mrqcof1End(void)
{
  int tid = blockIdx.x * blockDim.y + threadIdx.y;
  auto CUDA_LCC = &CUDA_CC[tid];

  if(__ldg(&isInvalid[tid])) return;
  if(!__ldg(&isNiter[tid])) return;
  if(!__ldg(&isAlamda[tid])) return;

  mrqcof_end(CUDA_LCC, CUDA_LCC->alpha);
  Ochisq[tid] = trial_chisqg[tid];
}

__global__
__launch_bounds__(768)
void CudaCalculateIter1Mrqcof2Start(void)
{
  int tid = blockIdx() * blockDim.x + threadIdx.x;

  if(tid < blockDim.y * gridDim.x)
    {
      //auto CUDA_LCC = &CUDA_CC[tid];
 
      double *a = atry[tid]; //CUDA_LCC->atry;
      blmatrix(a[CUDA_ma - CUDA_Nphpar - 4], a[CUDA_ma - CUDA_Nphpar - 3], tid);
    }

  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];
  
  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  if(threadIdx.x == 0)
    {
      trial_chisqg[bid] = 0;
      npg[bid] = 0;
      npg1[bid] = 0;
      npg2[bid] = 0;
      aveg[bid] = 0;
    }
  
  mrqcof_start(CUDA_LCC, atry[bid], CUDA_LCC->covar, CUDA_LCC->da, bid);
}

__global__ void CudaCalculateIter1Mrqcof2End(void)
{
  int tid = blockIdx.x * blockDim.y + threadIdx.y;
  auto CUDA_LCC = &CUDA_CC[tid];

  if(__ldg(&isInvalid[tid])) return;
  if(!__ldg(&isNiter[tid])) return;

  mrqcof_end(CUDA_LCC, CUDA_LCC->covar);
  Chisq[tid] = __ldg(&trial_chisqg[tid]);
}

__global__
__launch_bounds__(768) //768
void CudaCalculateIter2(void)
{
  //bool beenThere = false;
  int bid = blockIdx();
  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  
  int nf = CUDA_Numfac;
  auto CUDA_LCC = &CUDA_CC[bid];

  double chisq = __ldg(&Chisq[bid]);
  
  if(Niter[bid] == 1 || chisq < Ochisq[bid])
    {
      curv(CUDA_LCC, CUDA_LCC->cg, bid);
      
      double a[3] = {0, 0, 0};

      int j = threadIdx.x + 1;

      double const * __restrict__ areap = Areag[bid];
#pragma unroll 9
      while(j <= nf)
	{
	  double dd = areap[j];
#pragma unroll 3
	  for(int i = 0; i < 3; i++)
	    {
	      double const * __restrict__ norp = CUDA_Nor[i];
	      a[i] += dd * norp[j];
	    }
	  j += CUDA_BLOCK_DIM;
	}
      
#pragma unroll
      for(int off = CUDA_BLOCK_DIM/2; off > 0; off >>= 1)
	{
	  double b[3];
#pragma unroll 3
	  for(int i = 0; i < 3; i++)
	    b[i] = __shfl_down_sync(0xffffffff, a[i], off);
#pragma unroll 3
	  for(int i = 0; i < 3; i++)
	    a[i] += b[i];
	}
      
      //__syncwarp();
      if(threadIdx.x == 0)
	{
	  double conwr2 = CUDA_conw_r, aa = 0;
	  
	  Ochisq[bid] = chisq;
	  conwr2 *= conwr2;

#pragma unroll 3
	  for(int i = 0; i < 3; i++)
	    {
	      aa += a[i]*a[i];
	    }
	  
	  double rchisq = chisq - aa * conwr2; //(CUDA_conw_r * CUDA_conw_r);
	  double dev_old = dev_oldg[bid];
	  double dev_new = __dsqrt_rn(rchisq / (CUDA_ndata - 3));
	  chck[bid] = norm3d(a[0], a[1], a[2]);

	  dev_newg[bid]  = dev_new;
	  double diff    = dev_old - dev_new;
	  
	  /* 
	  // only if this step is better than the previous,
	  // 1e-10 is for numeric errors 
	  */
	  
	  if(diff > 1e-10)
	    {
	      iter_diffg[bid] = diff; 
	      dev_oldg[bid] = dev_new; 
	    }
	}
    }
}

__global__ void CudaCalculateFinishPole(void)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  auto CUDA_LCC = &CUDA_CC[tid];
  //auto CUDA_LFR = &CUDA_FR[tid];

  if(__ldg(&isInvalid[tid])) return;
  
  double dn = __ldg(&dev_newg[tid]);
  int nf = CUDA_Numfac;
  
  if(dn >= __ldg(&dev_best[tid]))
    return;

  double dark = __ldg(&chck[tid]); 

  register double tot = 0, tot2 = 0;
  double const * __restrict__ p = &(Areag[tid][1]); //??????????????????
#pragma unroll 4
  for(int i = 0; i < nf - 1; i++)
    {
      tot  += __ldca(p++);
      i++;
      tot2 += __ldca(p++);
    }
  if(nf & 1)
    tot += __ldca(p); //LDG_d_ca(CUDA_LCC->Area, (nf - 1));
  //tot += CUDA_LCC->Area[nf - 1];
  
  tot = __drcp_rn(tot + tot2);
  
  /* period solution */
  double period = 2.0 * PI * __drcp_rn(CUDA_LCC->cg[CUDA_Ncoef + 3]);

  /* pole solution */
  double la_tmp = RAD2DEG * CUDA_LCC->cg[CUDA_Ncoef + 2];
  double be_tmp = 90.0 - RAD2DEG * CUDA_LCC->cg[CUDA_Ncoef + 1];

  dev_best[tid] = dn;
  dark_best[tid] = dark * 100.0 * tot;
  per_best[tid] = period;
  la_best[tid] = la_tmp;
  be_best[tid] = be_tmp;
}

__global__ void CudaCalculateFinish(void)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  //  auto CUDA_LCC = &CUDA_CC[tid];
  //auto CUDA_LFR = &CUDA_FR[tid];

  if(__ldg(&isInvalid[tid])) return;

  double lla_best = la_best[tid];
  if(lla_best < 0)
    la_best[tid] = lla_best + 360;

  if(isnan(__ldg(&dark_best[tid])) == 1)
    dark_best[tid] = 1.0;
}
