#include "hip/hip_runtime.h"
#define SWAP(a,b) {temp=(a);(a)=(b);(b)=temp;}
#define SWAP4(a,b) {double x[4],y[4];for(int t1=0;t1<4;t1++) x[t1]=(a)[t1];for(int r1=0;r1<4;r1++) y[r1]=(b)[r1];for(int t2=0;t2<4;t2++)(b)[t2]=(x)[t2];for(int t3=0;t3<4;t3++)(a)[t3]=y[t3];}
#define SWAP8(a,b) {double x[8];for(int t1=0;t1<8;t1++) x[t1]=(a)[t1];for(int t2=0;t2<8;t2++)(a)[t2]=(b)[t2];for(int t3=0;t3<8;t3++)(b)[t3]=x[t3];}
#define SWAP4n(a,b,n) {double x[4],y[4];for(int t1=0;t1<4;t1++)x[t1]=(a)[t1*n];for(int r1=0;r1<4;r1++)y[r1]=(b)[r1*n];for(int t2=0;t2<4;t2++)(b)[t2*n]=x[t2];for(int t3=0;t3<4;t3++)(a)[t3*n]=y[t3];}
#define SWAP8n(a,b,n) {double x[8];for(int t1=0;t1<8;t1++)x[t1]=(a)[t1*n];for(int t2=0;t2<8;t2++)(a)[t2*n]=(b)[t2*n];for(int t3=0;t3<8;t3++)(b)[t3*n]=x[t3];}

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "globals_CUDA.h"
#include "declarations_CUDA.h"
#include <hip/hip_runtime.h>
#include <>

__device__ int __forceinline__ gauss_errc(freq_context * __restrict__ CUDA_LCC, int ma)
{
  __shared__ int16_t sh_icol[N80]; //[CUDA_BLOCK_DIM];
  __shared__ int16_t sh_irow[N80]; //[CUDA_BLOCK_DIM];
  __shared__ double sh_big[N80]; //[CUDA_BLOCK_DIM];
  __shared__ double pivinv;
  __shared__ int icol;

  __shared__ int16_t indxr[MAX_N_PAR + 1];
  __shared__ int16_t indxc[MAX_N_PAR + 1];
  __shared__ int16_t ipiv[MAX_N_PAR + 1];

  int mf1 = CUDA_mfit1;
  int i, licol = 0, irow = 0, j, k, l, ll;
  double big, dum, temp;
  int mf = CUDA_mfit;
  
  j = threadIdx.x + 1;

#pragma unroll 9
  while(j <= mf)
    {
      ipiv[j] = 0;
      j += CUDA_BLOCK_DIM;
    }

  __syncwarp();

  double * __restrict__ covarp = CUDA_LCC->covar;

#pragma unroll 1
  for(i = 1; i <= mf; i++)
    {
      big = 0.0;
      irow = 0;
      licol = 0;
      j = threadIdx.x + 1;

#pragma unroll 2
      while(j <= mf)
	{
	  if(ipiv[j] != 1)
	    {
	      int ixx = j * mf1 + 1;
#pragma unroll 4
	      for(k = 1; k <= mf; k++, ixx++)
		{
		  int ii = ipiv[k];
		  if(ii == 0)
		    {
		      double tmpcov = fabs(__ldg(&covarp[ixx]));
		      if(tmpcov >= big)
			{
			  irow = j;
			  licol = k;
			  big = tmpcov;
			}
		    }
		  else if(ii > 1)
		    {
		      return(1);
		    }
		}
	    }
	  j += CUDA_BLOCK_DIM;
	}
      //      sh_big[threadIdx.x] = big;
      //      sh_irow[threadIdx.x] = irow;
      //      sh_icol[threadIdx.x] = licol;
      j = threadIdx.x;
      while(j <= mf)
	{      
	  sh_big[j] = big;
	  sh_irow[j] = irow;
	  sh_icol[j] = licol;
	  j += CUDA_BLOCK_DIM;
	}
      
      __syncwarp();
      
      if(threadIdx.x == 0)
	{
	  big = sh_big[0];
	  icol = sh_icol[0];
	  irow = sh_irow[0];
#pragma unroll 2
	  for(j = 1; j <= mf; j++)
	    {
	      if(sh_big[j] >= big)
		{
		  big = sh_big[j];
		  irow = sh_irow[j];
		  icol = sh_icol[j];
		}
	    }
	  ++(ipiv[icol]);

	  double * __restrict__ dapp = CUDA_LCC->da;

	  if(irow != icol)
	    {
	      double * __restrict__ cvrp = covarp + irow * mf1; 
	      double * __restrict__ cvcp = covarp + icol * mf1; 
#pragma unroll 4
	      for(l = 1; l <= mf - 3; l += 4)
		{
		  SWAP4(cvrp, cvcp);
		  cvrp += 4;
		  cvcp += 4;
		}
	      
#pragma unroll 3
	      for(; l <= mf; l++)
		{
		  SWAP(cvrp[0], cvcp[0]);
		  cvrp++;
		  cvcp++;
		}
	      
	      SWAP(dapp[irow], dapp[icol]);
	      //SWAP(b[irow],b[icol])
	    }
	  //CUDA_LCC->indxr[i] = irow;
	  indxr[i] = irow;
	  //CUDA_LCC->indxc[i] = icol;
	  indxc[i] = icol;
	  double cov = covarp[icol * mf1 + icol];
	  if(cov == 0.0) 
	    {
	      int bid = blockIdx();
	      j = 0;
	      
	      int    const * __restrict__ iap = CUDA_ia + 1;
	      double * __restrict__ atp = atry[bid] + 1; //CUDA_LCC->atry + 1;
	      double * __restrict__ cgp = CUDA_LCC->cg + 1;
	      double * __restrict__ dap = dapp;
#pragma unroll 4
	      for(int l = 1; l <= ma; l++)
		{
		  if(*iap)
		    {
		      dap++;
		      __stwb(atp,  *cgp + *dap);
		    }
		  iap++;
		  atp++;
		  cgp++;
		}
	      
	      return(2);
	    }
	  pivinv = ___drcp_rn(cov);
	  covarp[icol * mf1 + icol] = 1.0;
	  dapp[icol] *= pivinv;
	}
      
      __syncwarp();
      
      int x = threadIdx.x + 1;
      double * __restrict__ p = &covarp[icol * mf1];
#pragma unroll 2
      while(x <= mf)
	{
	  //if(x != 0)
	  __stwb(&p[x], __ldg(&p[x]) * pivinv);
	  x += CUDA_BLOCK_DIM;
	}
      
      __syncwarp();
      
#pragma unroll 2
      for(ll = 1; ll <= mf; ll++)
	if(ll != icol)
	  {
	    int ixx = ll * mf1, jxx = icol * mf1;
	    dum = __ldg(&covarp[ixx + icol]);
	    covarp[ixx + icol] = 0.0;
	    ixx++;
	    jxx++;
	    ixx += threadIdx.x;
	    jxx += threadIdx.x;
	    l = threadIdx.x + 1;
#pragma unroll 2
	    while(l <= mf)
	      {
		__stwb(&covarp[ixx],  __ldg(&covarp[ixx]) - __ldg(&covarp[jxx]) * dum);
		l += CUDA_BLOCK_DIM;
		ixx += CUDA_BLOCK_DIM;
		jxx += CUDA_BLOCK_DIM;
	      }
	    double *dapp = CUDA_LCC->da;
	    __stwb(&dapp[ll], __ldg(&dapp[ll]) - __ldg(&dapp[icol]) * dum);
	  }
      
      __syncwarp();
    }

  l = mf - threadIdx.x;

  while(l >= 1)
    {
      //int r = CUDA_LCC->indxr[l];
      int r = indxr[l];
      //int c = CUDA_LCC->indxc[l];
      int c = indxc[l];
      if(r != c)
	{
	  double * __restrict__ cvp1 = &(covarp[0]), * __restrict__ cvp2;
	  cvp2 = cvp1;
	  int i1 = mf1 + r;
	  int i2 = mf1 + c;
	  cvp1 = cvp1 + i1;
	  cvp2 = cvp2 + i2;
#pragma unroll 4
	  for(k = 1; k <= mf - 3; k += 4)
	    {
	      SWAP4n(cvp1, cvp2, mf1);
	      cvp1 += mf1 * 4;
	      cvp2 += mf1 * 4;
	    }
#pragma unroll 3
	  for(; k <= mf; k++)
	    {
	      SWAP(cvp1[0], cvp2[0]);
	      cvp1 += mf1;
	      cvp2 += mf1;
	    }
	}
      l -= CUDA_BLOCK_DIM;
    }

  __syncwarp();

  return(0);
}
#undef SWAP