#include "hip/hip_runtime.h"
/* slighly changed code from Numerical Recipes
   converted from Mikko's fortran code

   8.11.2006
*/

#include <stdio.h>
#include <stdlib.h>
#include "globals_CUDA.h"
#include "declarations_CUDA.h"
#include <>



__device__ void __forceinline__ mrqcof_start(freq_context * __restrict__ CUDA_LCC,
											 double * __restrict__ a,
											 double * __restrict__ alpha,
											 double * __restrict__ beta,
											 int bid)
{
	int j, k;
	int mf = CUDA_mfit, mf1 = CUDA_mfit1;

	/* N.B. curv and blmatrix called outside bright
	   because output same for all points */
	curv(CUDA_LCC, a, bid);

#pragma unroll 4
	for (j = 1; j <= mf; j++)
	{
		alpha += mf1;
		k = threadIdx.x + 1;
#pragma unroll
		while (k <= j)
		{
			__stwb(&alpha[k], 0.0);
			k += CUDA_BLOCK_DIM;
		}
	}

	j = threadIdx.x + 1;
#pragma unroll 2
	while (j <= mf)
	{
		__stwb(&beta[j], 0.0);
		j += CUDA_BLOCK_DIM;
	}

	// __syncthreads(); //pro jistotu
}

__device__ double __forceinline__ mrqcof_end(freq_context * __restrict__ CUDA_LCC, double * __restrict__ alpha)
{
	int j, k, mf = CUDA_mfit, mf1 = CUDA_mfit1;
	int tid = threadIdx.x;
	double *__restrict__ app = alpha + mf1 + 2 + tid;
	;
	double const *__restrict__ ap2 = alpha + (2 + tid) * mf1;
	long int mf1add = sizeof(double) * mf1;
#pragma unroll
	for (j = 2 + tid; j <= mf; j += blockDim.x)
	{
		double *__restrict__ ap = app;
#pragma unroll
		for (k = 1; k <= j - 1; k++)
		{
			__stwb(ap, __ldca(&ap2[k]));
			// ap  += mf1;
			ap = (double *)(((char *)ap) + mf1add);
		}
		app += blockDim.x;
		// ap2 += mf1;
		ap2 = (double *)(((char *)ap2) + mf1add * blockDim.x);
	}

	return 0; // trial_chisqg[bid];
}

__device__ void __forceinline__ mrqcof_matrix(freq_context *__restrict__ CUDA_LCC,
											  double *__restrict__ a,
											  int Lpoints, int bid)
{
	matrix_neo(CUDA_LCC, a, npg[bid], Lpoints, bid);
}

__device__ void __forceinline__ mrqcof_curve1(freq_context *__restrict__ CUDA_LCC,
											  double *__restrict__ a,
											  int Inrel, int Lpoints, int bid)
{
	int lnp, Lpoints1 = Lpoints + 1;
	double lave = 0;

	int n = threadIdx.x;
	if (Inrel == 1)
	{
#pragma unroll 1
		while (n <= Lpoints)
		{
			bright(CUDA_LCC, a, n, Lpoints1, 1); // jp <-- n, OK, consecutive
			n += CUDA_BLOCK_DIM;
		}
	}

	int ma = CUDA_ma;

	__syncwarp();
	double *__restrict__ dytemp = CUDA_LCC->dytemp, *__restrict__ ytemp = CUDA_LCC->ytemp;

	if (Inrel == 1)
	{
		double const *__restrict__ pp = &(dytemp[2 * Lpoints1 + threadIdx.x + 1]); // good, consecutive
		int bid = blockIdx();
#pragma unroll 1
		for (int i = 2; i <= ma; i++)
		{
			double dl = 0, dl2 = 0;
			int nn = threadIdx.x + 1;
			double const *__restrict__ p = pp;

#pragma unroll 2
			while (nn <= Lpoints - CUDA_BLOCK_DIM)
			{
				dl += p[0];
				dl2 += p[CUDA_BLOCK_DIM];
				p += 2 * CUDA_BLOCK_DIM;
				nn += 2 * CUDA_BLOCK_DIM;
			}
			// #pragma unroll 1
			if (nn <= Lpoints)
			{
				dl += p[0];
				// p  += CUDA_BLOCK_DIM;
				// nn += CUDA_BLOCK_DIM;
			}

			dl += dl2;

			dl += __shfl_down_sync(0xffffffff, dl, 16);
			dl += __shfl_down_sync(0xffffffff, dl, 8);
			dl += __shfl_down_sync(0xffffffff, dl, 4);
			dl += __shfl_down_sync(0xffffffff, dl, 2);
			dl += __shfl_down_sync(0xffffffff, dl, 1);

			pp += Lpoints1;

			if (threadIdx.x == 0)
				dave[bid][i] = dl;
		}

		double d = 0, d2 = 0;
		int n = threadIdx.x + 1;
		double const *__restrict__ p2 = &(ytemp[n]);

#pragma unroll 2
		while (n <= Lpoints - CUDA_BLOCK_DIM)
		{
			d += p2[0];
			d2 += p2[CUDA_BLOCK_DIM];
			p2 += 2 * CUDA_BLOCK_DIM;
			n += 2 * CUDA_BLOCK_DIM;
		}

		if (n <= Lpoints)
		{
			d += p2[0];
		}
		d += d2;

		d += __shfl_down_sync(0xffffffff, d, 16);
		d += __shfl_down_sync(0xffffffff, d, 8);
		d += __shfl_down_sync(0xffffffff, d, 4);
		d += __shfl_down_sync(0xffffffff, d, 2);
		d += __shfl_down_sync(0xffffffff, d, 1);

		lave = d;
	}

	if (threadIdx.x == 0)
	{
		lnp = npg[bid];
		aveg[bid] = lave;
		npg[bid] = lnp + Lpoints;
	}
	__syncwarp();
}

// __device__ void mrqcof_curve1_last(freq_context *CUDA_LCC, double a[],
// 	      double *alpha, double beta[],int Inrel,int Lpoints)
// {
// 	int l,jp, lnp;
//    double ymod, lave;

//    lnp=(*CUDA_LCC).np;
//    //
//    if (threadIdx.x==0)
//    {
// 	   if (Inrel == 1) /* is the LC relative? */
// 	   {
// 		  lave = 0;
// 		  for (l = 1; l <= CUDA_ma; l++)
// 		  (*CUDA_LCC).dave[l]=0;
// 	   }
// 	   else
// 		  lave=(*CUDA_LCC).ave;
//    }
// //precalc thread boundaries
//     int tmph,tmpl;
// 	tmph=CUDA_ma/CUDA_BLOCK_DIM;
// 	if(CUDA_ma%CUDA_BLOCK_DIM) tmph++;
// 	tmpl=threadIdx.x*tmph;
// 	tmph=tmpl+tmph;
// 	if (tmph>CUDA_ma) tmph=CUDA_ma;
// 	tmpl++;
// //
//     int brtmph,brtmpl;
// 	brtmph=CUDA_Numfac/CUDA_BLOCK_DIM;
// 	if(CUDA_Numfac%CUDA_BLOCK_DIM) brtmph++;
// 	brtmpl=threadIdx.x*brtmph;
// 	brtmph=brtmpl+brtmph;
// 	if (brtmph>CUDA_Numfac) brtmph=CUDA_Numfac;
// 	brtmpl++;

// 	__syncthreads();

//       for (jp = 1; jp <= Lpoints; jp++)
//       {
//          lnp++;

//          ymod = conv(CUDA_LCC,jp-1,tmpl,tmph,brtmpl,brtmph);

// 		 if (threadIdx.x==0)
// 		 {
// 			 (*CUDA_LCC).ytemp[jp] = ymod;

// 			 if (Inrel == 1)
// 				lave = lave + ymod;
// 		 }
// 		for (l=tmpl; l <= tmph; l++)
// 		{
// 			(*CUDA_LCC).dytemp[jp+l*(Lpoints+1)] = (*CUDA_LCC).dyda[l];
// 			if (Inrel == 1)
// 				(*CUDA_LCC).dave[l] = (*CUDA_LCC).dave[l] + (*CUDA_LCC).dyda[l];
// 		}
// 		/* save lightcurves */
// 		 __syncthreads();

// /*         if ((*CUDA_LCC).Lastcall == 1) always ==0
// 			 (*CUDA_LCC).Yout[np] = ymod;*/
//       } /* jp, lpoints */
// 	 if (threadIdx.x==0)
// 	 {
// 		  (*CUDA_LCC).np=lnp;
// 		  (*CUDA_LCC).ave=lave;
// 	 }
// }

__device__ void __forceinline__ mrqcof_curve1_lastI1(
	freq_context * __restrict__ CUDA_LCC,
	double * __restrict__ a,
	double * __restrict__ alpha,
	double * __restrict__ beta,
	int bid)
{
	int Lpoints = 3;
	int Lpoints1 = Lpoints + 1;
	int jp, lnp;
	double ymod, lave;
	__shared__ double dyda[BLOCKX4][N80];
	double *__restrict__ dydap = dyda[threadIdx.y];
	// int bid = blockIdx();

	lnp = npg[bid];

	int n = threadIdx.x + 1, ma = CUDA_ma;
	double *__restrict__ p = &(dave[bid][n]);
#pragma unroll 2
	while (n <= ma)
	{
		*p = 0;
		p += CUDA_BLOCK_DIM;
		n += CUDA_BLOCK_DIM;
	}
	lave = 0;

	//__syncthreads();

	double *__restrict__ dytemp = CUDA_LCC->dytemp, *ytemp = CUDA_LCC->ytemp;
	long int lpadd = sizeof(double) * Lpoints1;

#pragma unroll 1
	for (jp = 1; jp <= Lpoints; jp++)
	{
		ymod = conv(CUDA_LCC, (jp - 1), dydap, bid);

		lnp++;

		if (threadIdx.x == 0)
		{
			ytemp[jp] = ymod;
			lave = lave + ymod;
		}

		int n = threadIdx.x + 1;
		double const *__restrict__ a;
		double *__restrict__ b, *__restrict__ c;

		a = &(dydap[n - 1]);
		b = &(dave[bid][n]);
#ifdef DYTEMP_NEW
		// c = &(dytemp2[blockIdx()][jp][n]);
#else
		c = &(dytemp[jp + Lpoints1 * n]); // ZZZ bad store order, strided
#endif
		// unrl2
#pragma unroll 2
		while (n <= ma - CUDA_BLOCK_DIM)
		{
			double d = a[0], bb = b[0];
			double d2 = a[CUDA_BLOCK_DIM], bb2 = b[CUDA_BLOCK_DIM];
#ifdef DYTEMP_NEW
			dytemp2[bid][jp][n] = d;
#else
			c[0] = d;
#endif
			// c += Lpoints1;
			c = (double *)(((char *)c) + lpadd);
			b[0] = bb + d;
#ifdef DYTEMP_NEW
			dytemp2[bid][jp][n + CUDA_BLOCK_DIM] = d2;
#else
			c[0] = d2;
#endif
			// c += Lpoints1;
			c = (double *)(((char *)c) + lpadd);
			b[CUDA_BLOCK_DIM] = bb2 + d2;
			a += 2 * CUDA_BLOCK_DIM;
			b += 2 * CUDA_BLOCK_DIM;
			n += 2 * CUDA_BLOCK_DIM;
		}
		// #pragma unroll 1
		if (n <= ma)
		{
			double d = a[0], bb = b[0];
#ifdef DYTEMP_NEW
			dytemp2[bid][jp][n] = d;
#else
			c[0] = d;
#endif
			b[0] = bb + d;
		}
	} /* jp, lpoints */

	if (threadIdx.x == 0)
	{
		npg[bid] = lnp;
		aveg[bid] = lave;
	}

	/* save lightcurves */
	__syncwarp();
}

__device__ void __forceinline__ mrqcof_curve1_lastI0(freq_context * __restrict__ CUDA_LCC,
													 double * __restrict__ a,
													 double * __restrict__ alpha,
													 double * __restrict__ beta,
													 int bid)
{
	int Lpoints = 3;
	int Lpoints1 = Lpoints + 1;
	int jp, lnp;
	double ymod;
	__shared__ double dyda[BLOCKX4][N80];
	// int bid = blockIdx();
	double *__restrict__ dydap = dyda[threadIdx.y];

	lnp = npg[bid];

	//  if(threadIdx.x == 0)
	//  lave = CUDA_LCC->ave;

	//__syncthreads();

	int ma = CUDA_ma;
	double *__restrict__ dytemp = CUDA_LCC->dytemp, *ytemp = CUDA_LCC->ytemp;

#pragma unroll 3
	for (jp = 1; jp <= Lpoints; jp++)
	{
		lnp++;

		ymod = conv(CUDA_LCC, (jp - 1), dydap, bid);

		if (threadIdx.x == 0)
			ytemp[jp] = ymod;

		int n = threadIdx.x + 1;
		double *__restrict__ p = &dytemp[jp + Lpoints1 * n]; // ZZZ bad store order, strided
#pragma unroll 2
		while (n <= ma - CUDA_BLOCK_DIM)
		{
			double d = dydap[n - 1];
			double d2 = dydap[n + CUDA_BLOCK_DIM - 1];
#ifdef DYTEMP_NEW
			dytemp2[bid][jp][n] = d;
#else
			*p = d; //  YYYY
#endif
			p += Lpoints1 * CUDA_BLOCK_DIM;
#ifdef DYTEMP_NEW
			dytemp2[bid][jp][n + CUDA_BLOCK_DIM] = d2;
#else
			*p = d2;
#endif
			p += Lpoints1 * CUDA_BLOCK_DIM;
			n += 2 * CUDA_BLOCK_DIM;
		}
		// #pragma unroll 1
		if (n <= ma)
		{
			double d = dydap[n - 1];
#ifdef DYTEMP_NEW
			dytemp2[bid][jp][n] = d;
#else
			*p = d;
#endif
			// p += Lpoints1 * CUDA_BLOCK_DIM;
			// n += CUDA_BLOCK_DIM;
		}
	} /* jp, lpoints */

	if (threadIdx.x == 0)
	{
		npg[bid] = Lpoints; // lnp;
		//      CUDA_LCC->ave = lave;
	}

	/* save lightcurves */
	//__syncthreads();
}