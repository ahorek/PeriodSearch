#include "hip/hip_runtime.h"
/* computes integrated brightness of all visible and iluminated areas
   and its derivatives

   8.11.2006
*/

#include <cmath>
#include "globals_CUDA.h"
#include <>

__device__ void __forceinline__ matrix_neo(freq_context *__restrict__ CUDA_LCC, double const *__restrict__ cg, int lnp1, int Lpoints, int bid)
{
  int lnp, jp;
  int blockidx = bid;

  jp = threadIdx.x + 1;

  double nc02r = ___drcp_rn(cg[CUDA_ncoef0 + 2]);
  double phi0 = CUDA_Phi_0;
  double nc02r2 = nc02r * nc02r;

#pragma unroll 1
  while (jp <= Lpoints)
  {
    double f, cf, sf, pom, pom0, alpha;
    double ee_1, ee_2, ee_3, ee0_1, ee0_2, ee0_3, t, tmat1, tmat2, tmat3;

    lnp = lnp1 + jp;

    ee_1 = CUDA_ee[0][lnp]; // position vectors
    ee0_1 = CUDA_ee0[0][lnp];
    ee_2 = CUDA_ee[1][lnp];
    ee0_2 = CUDA_ee0[1][lnp];
    ee_3 = CUDA_ee[2][lnp];
    ee0_3 = CUDA_ee0[2][lnp];
    t = CUDA_tim[lnp];
    double nc00 = cg[CUDA_ncoef0 + 0];

    alpha = acos(((ee_1 * ee0_1) + ee_2 * ee0_2) + ee_3 * ee0_3);
    f = nc00 * t + phi0;

    /* Exp-lin model (const.term=1.) */
    double ff = exp2(-1.44269504088896 * (alpha * nc02r));

    double nc01 = cg[CUDA_ncoef0 + 1];
    double nc03 = cg[CUDA_ncoef0 + 3];

    /* fmod may give little different results than Mikko's */
    f = f - 2.0 * PI * round(f * (1.0 / (2.0 * PI))); // 3:41.9

    double scale = 1.0 + nc01 * ff + nc03 * alpha;
    double d2 = nc01 * ff * alpha * nc02r2;

    //  matrix start

    __builtin_assume(f > (-2.0 * PI) && f < (2.0 * PI));
    sincos(f, &sf, &cf);

    CUDA_scale[blockidx][jp] = scale;

    jp_dphp[0][blockidx][jp] = ff;
    jp_dphp[1][blockidx][jp] = d2;
    jp_dphp[2][blockidx][jp] = alpha;

    /* rotation matrix, Z axis, angle f */

    double Blmat00 = __ldg(&Blmat[0][0][blockidx]);
    double Blmat10 = __ldg(&Blmat[1][0][blockidx]);
    double Blmat01 = __ldg(&Blmat[0][1][blockidx]);
    double Blmat11 = __ldg(&Blmat[1][1][blockidx]);
    double Blmat02 = __ldg(&Blmat[0][2][blockidx]);

    tmat1 = cf * Blmat00;
    tmat2 = cf * Blmat01;
    tmat3 = cf * Blmat02;
    tmat1 += sf * Blmat10;
    tmat2 += sf * Blmat11;
    pom = tmat1 * ee_1;
    pom0 = tmat1 * ee0_1;
    pom += tmat2 * ee_2;
    pom0 += tmat2 * ee0_2;
    pom += tmat3 * ee_3;
    pom0 += tmat3 * ee0_3;
    double pom1 = pom;
    double pom1_0 = pom0;
    double pom1_t = -t * pom;
    double pom1_t0 = -t * pom0;
    ge[0][0][blockidx][jp] = pom1;
    ge[1][0][blockidx][jp] = pom1_0;
    gde[0][1][2][blockidx][jp] = pom1_t;
    gde[1][1][2][blockidx][jp] = pom1_t0;

    double msf = -sf;

    tmat1 = msf * Blmat00;
    tmat2 = msf * Blmat01;
    tmat3 = msf * Blmat02;
    tmat1 += cf * Blmat10;
    tmat2 += cf * Blmat11;
    pom = tmat1 * ee_1;
    pom0 = tmat1 * ee0_1;
    pom += tmat2 * ee_2;
    pom0 += tmat2 * ee0_2;
    pom += tmat3 * ee_3;
    pom0 += tmat3 * ee0_3;
    double pom2 = pom;
    double pom2_0 = pom0;
    double pom2_t = t * pom;
    double pom2_t0 = t * pom0;
    ge[0][1][blockidx][jp] = pom2;
    ge[1][1][blockidx][jp] = pom2_0;
    gde[0][0][2][blockidx][jp] = pom2_t;
    gde[1][0][2][blockidx][jp] = pom2_t0;

    tmat1 = __ldg(&Blmat[2][0][blockidx]);
    tmat2 = __ldg(&Blmat[2][1][blockidx]);
    tmat3 = __ldg(&Blmat[2][2][blockidx]);

    pom = tmat1 * ee_1;
    pom0 = tmat1 * ee0_1;
    pom += tmat2 * ee_2;
    pom0 += tmat2 * ee0_2;
    pom += tmat3 * ee_3;
    pom0 += tmat3 * ee0_3;

    double Dblm000 = __ldg(&Dblm[0][0][0][blockidx]);
    double Dblm001 = __ldg(&Dblm[0][0][1][blockidx]);
    double Dblm002 = __ldg(&Dblm[0][0][2][blockidx]);

    ge[0][2][blockidx][jp] = pom;
    ge[1][2][blockidx][jp] = pom0;

    tmat1 = cf * Dblm000;
    tmat2 = cf * Dblm001;
    tmat3 = cf * Dblm002;

    pom = tmat1 * ee_1;
    pom0 = tmat1 * ee0_1;
    pom += tmat2 * ee_2;
    pom0 += tmat2 * ee0_2;
    pom += tmat3 * ee_3;
    pom0 += tmat3 * ee0_3;
    gde[0][0][0][blockidx][jp] = pom;
    gde[1][0][0][blockidx][jp] = pom0;

    tmat1 = msf * Dblm000;
    tmat2 = msf * Dblm001;
    tmat3 = msf * Dblm002;
    pom = tmat1 * ee_1;
    pom0 = tmat1 * ee0_1;
    pom += tmat2 * ee_2;
    pom0 += tmat2 * ee0_2;
    pom += tmat3 * ee_3;
    pom0 += tmat3 * ee0_3;

    double Dblm100 = __ldg(&Dblm[1][0][0][blockidx]);
    double Dblm101 = __ldg(&Dblm[1][0][1][blockidx]);
    double Dblm110 = __ldg(&Dblm[1][1][0][blockidx]);
    double Dblm111 = __ldg(&Dblm[1][1][1][blockidx]);

    gde[0][1][0][blockidx][jp] = pom;
    gde[1][1][0][blockidx][jp] = pom0;

    tmat1 = cf * Dblm100;
    tmat2 = cf * Dblm101;
    tmat1 += sf * Dblm110;
    tmat2 += sf * Dblm111;

    pom = tmat1 * ee_1;
    pom0 = tmat1 * ee0_1;
    pom += tmat2 * ee_2;
    pom0 += tmat2 * ee0_2;

    tmat1 = msf * Dblm100 + cf * Dblm110;
    tmat2 = msf * Dblm101 + cf * Dblm111;

    gde[0][0][1][blockidx][jp] = pom;
    gde[1][0][1][blockidx][jp] = pom0;

    pom = tmat1 * ee_1;
    pom0 = tmat1 * ee0_1;
    pom += tmat2 * ee_2;
    pom0 += tmat2 * ee0_2;

    double Dblm020 = __ldg(&Dblm[0][2][0][blockidx]);
    double Dblm021 = __ldg(&Dblm[0][2][1][blockidx]);
    double Dblm022 = __ldg(&Dblm[0][2][2][blockidx]);

    gde[0][1][1][blockidx][jp] = pom;
    gde[1][1][1][blockidx][jp] = pom0;

    tmat1 = Dblm020;
    tmat2 = Dblm021;
    tmat3 = Dblm022;

    pom = tmat1 * ee_1;
    pom0 = tmat1 * ee0_1;
    pom += tmat2 * ee_2;
    pom0 += tmat2 * ee0_2;
    pom += tmat3 * ee_3;
    pom0 += tmat3 * ee0_3;

    double Dblm120 = __ldg(&Dblm[1][2][0][blockidx]);
    double Dblm121 = __ldg(&Dblm[1][2][1][blockidx]);

    gde[0][2][0][blockidx][jp] = pom;
    gde[1][2][0][blockidx][jp] = pom0;

    tmat1 = Dblm120;
    tmat2 = Dblm121;

    pom = tmat1 * ee_1;
    pom0 = tmat1 * ee0_1;
    pom += tmat2 * ee_2;
    pom0 += tmat2 * ee0_2;

    gde[0][2][2][blockidx][jp] = 0;
    gde[1][2][2][blockidx][jp] = 0;

    gde[0][2][1][blockidx][jp] = pom;
    gde[1][2][1][blockidx][jp] = pom0;

    jp += CUDA_BLOCK_DIM;
  }
  __syncwarp();
}

__device__ double __forceinline__ bright(freq_context *__restrict__ CUDA_LCC,
                                         double *__restrict__ cg,
                                         int jp /*threadIdx, ok!*/, int Lpoints1, int Inrel)
{
  int ncoef0, ncoef, incl_count = 0;
  int i, j, blockidx = blockIdx();
  double cl, cls, dnom, s, Scale;
  double e_1, e_2, e_3, e0_1, e0_2, e0_3;
  double de[3][3], de0[3][3];

  ncoef0 = CUDA_ncoef0; // ncoef - 2 - CUDA_Nphpar;
  ncoef = CUDA_ma;
  cl = exp(cg[ncoef - 1]); /* Lambert */
  cls = cg[ncoef];         /* Lommel-Seeliger */

  /* matrix from neo */
  /* derivatives */

  e_1 = __ldg(&ge[0][0][blockidx][jp]);
  e_2 = __ldg(&ge[0][1][blockidx][jp]);
  e_3 = __ldg(&ge[0][2][blockidx][jp]);
  e0_1 = __ldg(&ge[1][0][blockidx][jp]);
  e0_2 = __ldg(&ge[1][1][blockidx][jp]);
  e0_3 = __ldg(&ge[1][2][blockidx][jp]);

  de[0][0] = __ldg(&gde[0][0][0][blockidx][jp]);
  de[0][1] = __ldg(&gde[0][0][1][blockidx][jp]);
  de[0][2] = __ldg(&gde[0][0][2][blockidx][jp]);
  de[1][0] = __ldg(&gde[0][1][0][blockidx][jp]);
  de[1][1] = __ldg(&gde[0][1][1][blockidx][jp]);
  de[1][2] = __ldg(&gde[0][1][2][blockidx][jp]);
  de[2][0] = __ldg(&gde[0][2][0][blockidx][jp]);
  de[2][1] = __ldg(&gde[0][2][1][blockidx][jp]);
  de[2][2] = 0; // CUDA_LCC->de[2][2][jp];

  de0[0][0] = __ldg(&gde[1][0][0][blockidx][jp]);
  de0[0][1] = __ldg(&gde[1][0][1][blockidx][jp]);
  de0[0][2] = __ldg(&gde[1][0][2][blockidx][jp]);
  de0[1][0] = __ldg(&gde[1][1][0][blockidx][jp]);
  de0[1][1] = __ldg(&gde[1][1][1][blockidx][jp]);
  de0[1][2] = __ldg(&gde[1][1][2][blockidx][jp]);
  de0[2][0] = __ldg(&gde[1][2][0][blockidx][jp]);
  de0[2][1] = __ldg(&gde[1][2][1][blockidx][jp]);
  de0[2][2] = 0; // CUDA_LCC->de0[2][2][jp];

  /* Directions (and ders.) in the rotating system */

  //
  /*Integrated brightness (phase coeff. used later) */
  double lmu, lmu0, dsmu, dsmu0, sum1, sum10, sum2, sum20, sum3, sum30;
  double br, ar, tmp1, tmp2, tmp3, tmp4, tmp5;
  //   short int *incl=&CUDA_LCC->incl[threadIdx.x*MAX_N_FAC];
  //   double *dbr=&CUDA_LCC->dbr[threadIdx.x*MAX_N_FAC];

  short int incl[MAX_N_FAC];
  double dbr[MAX_N_FAC];
  // int2 bfr;
  int nf = CUDA_Numfac, nf1 = CUDA_Numfac1;

  int bid = blockidx;
  br = 0;
  tmp1 = 0;
  tmp2 = 0;
  tmp3 = 0;
  tmp4 = 0;
  tmp5 = 0;
  j = bid * nf1 + 1;
  double const *__restrict__ norp0;
  double const *__restrict__ norp1;
  double const *__restrict__ norp2;
  double const *__restrict__ areap;
  double const *__restrict__ dareap;
  norp0 = CUDA_Nor[0];
  norp1 = CUDA_Nor[1];
  norp2 = CUDA_Nor[2];
  // areap = CUDA_Area;
  areap = &(Areag[bid][0]);
  dareap = CUDA_Darea;

#pragma unroll 1
  for (i = 1; i <= nf && i <= MAX_N_FAC; i++, j++)
  {
    double n0 = norp0[i], n1 = norp1[i], n2 = norp2[i];
    lmu = e_1 * n0 + e_2 * n1 + e_3 * n2;
    lmu0 = e0_1 * n0 + e0_2 * n1 + e0_3 * n2;
    // if((lmu > TINY) && (lmu0 > TINY))
    //{
    if ((lmu <= TINY) || (lmu0 <= TINY))
      continue;
    dnom = lmu + lmu0;
    ar = __ldca(&areap[i]);

    double dnom_1 = ___drcp_rn(dnom);

    s = lmu * lmu0 * (cl + cls * dnom_1);
    double lmu0_dnom = lmu0 * dnom_1;

    br += ar * s;
    //
    dbr[incl_count] = __ldca(&dareap[i]) * s;
    incl[incl_count] = i;
    incl_count++;

    double lmu_dnom = lmu * dnom_1;
    dsmu = cls * (lmu0_dnom * lmu0_dnom) + cl * lmu0;
    dsmu0 = cls * (lmu_dnom * lmu_dnom) + cl * lmu;
    //	  double n0 = CUDA_Nor[0][i], n1 = CUDA_Nor[1][i], n2 = CUDA_Nor[2][i];

    sum1 = n0 * de[0][0] + n1 * de[1][0] + n2 * de[2][0];
    sum10 = n0 * de0[0][0] + n1 * de0[1][0] + n2 * de0[2][0];
    sum2 = n0 * de[0][1] + n1 * de[1][1] + n2 * de[2][1];
    sum20 = n0 * de0[0][1] + n1 * de0[1][1] + n2 * de0[2][1];
    sum3 = n0 * de[0][2] + n1 * de[1][2];    // + n2 * de[2][2];
    sum30 = n0 * de0[0][2] + n1 * de0[1][2]; // + n2 * de0[2][2];

    tmp1 += ar * (dsmu * sum1 + dsmu0 * sum10);
    tmp2 += ar * (dsmu * sum2 + dsmu0 * sum20);
    tmp3 += ar * (dsmu * sum3 + dsmu0 * sum30);

    tmp4 += ar * lmu * lmu0;
    tmp5 += ar * lmu * lmu0 * dnom_1; // lmu0 * __drcp_rn(lmu + lmu0);
    //}
  }

  // Scale = CUDA_LCC->jp_Scale[jp];
  Scale = __ldg(&CUDA_scale[bid][jp]);
  i = jp + (ncoef0 - 3 + 1) * Lpoints1;
#ifndef NEWDYTMP
  double *__restrict__ dytempp = CUDA_LCC->dytemp, *__restrict__ ytemp = CUDA_LCC->ytemp;
#else
  double *__restrict__ dytempp = dytemp[jp][0][bid], *__restrict__ ytemp = CUDA_LCC->ytemp;
#endif
  /* Ders. of brightness w.r.t. rotation parameters */
  dytempp[i] = Scale * tmp1;
  i += Lpoints1;
  dytempp[i] = Scale * tmp2;
  i += Lpoints1;
  dytempp[i] = Scale * tmp3;
  i += Lpoints1;

  /* Ders. of br. w.r.t. phase function params. */
  dytempp[i] = br * __ldg(&jp_dphp[0][bid][jp]);
  i += Lpoints1;
  dytempp[i] = br * __ldg(&jp_dphp[1][bid][jp]);
  i += Lpoints1;
  dytempp[i] = br * __ldg(&jp_dphp[2][bid][jp]);

  /* Ders. of br. w.r.t. cl, cls */
  dytempp[jp + (ncoef) * (Lpoints1)-Lpoints1] = Scale * tmp4 * cl;
  dytempp[jp + (ncoef) * (Lpoints1)] = Scale * tmp5;

  /* Scaled brightness */
  ytemp[jp] = br * Scale;

  ncoef0 -= 3;
  int m, m1, mr, iStart;
  int d, d1, dr;

  iStart = Inrel + 1;
  m = bid * CUDA_Dg_block + iStart * nf1;
  d = jp + Inrel * 2 * Lpoints1;

  m1 = m + nf1;
  mr = 2 * nf1;
  d1 = d + Lpoints1;
  dr = 2 * Lpoints1;

  /* Derivatives of brightness w.r.t. g-coeffs */
  if (incl_count)
  {
    double const *__restrict__ pCUDA_Dg = CUDA_Dg + m;
    double const *__restrict__ pCUDA_Dg1 = CUDA_Dg + m1;

#pragma unroll 1
    for (i = iStart; i <= ncoef0; i += 2, /*m += mr, m1 += mr,*/ d += dr, d1 += dr)
    {
      double tmp = 0, tmp1 = 0;

      if ((i + 1) <= ncoef0)
      {
#pragma unroll 2
        for (j = 0; j < incl_count - (UNRL - 1); j += UNRL)
        {
          double l_dbr[UNRL], l_tmp[UNRL], l_tmp1[UNRL];
          int l_incl[UNRL], ii;

          for (ii = 0; ii < UNRL; ii++)
          {
            l_incl[ii] = incl[j + ii];
            l_dbr[ii] = dbr[j + ii];
          }
          for (ii = 0; ii < UNRL; ii++)
          {
            l_tmp[ii] = pCUDA_Dg[l_incl[ii]];
            l_tmp1[ii] = pCUDA_Dg1[l_incl[ii]];
          }
          for (ii = 0; ii < UNRL; ii++)
          {
            double qq = l_dbr[ii];
            tmp += qq * l_tmp[ii];
            tmp1 += qq * l_tmp1[ii];
          }
        }
#pragma unroll 3
        for (; j < incl_count; j++)
        {
          int l_incl = incl[j];
          double l_dbr = dbr[j];
          double v1 = pCUDA_Dg[l_incl];
          double v2 = pCUDA_Dg1[l_incl];

          tmp += l_dbr * v1;
          tmp1 += l_dbr * v2;
        }
        __stwb(&dytempp[d], Scale * tmp);
        __stwb(&dytempp[d1], Scale * tmp1);
      }
      else
      {
#pragma unroll 2
        for (j = 0; j < incl_count - (UNRL - 1); j += UNRL)
        {
          double l_dbr[UNRL], l_tmp[UNRL];
          int l_incl[UNRL], ii;

          for (ii = 0; ii < UNRL; ii++)
          {
            l_incl[ii] = incl[j + ii];
          }

          for (ii = 0; ii < UNRL; ii++)
          {
            l_dbr[ii] = dbr[j + ii];
            l_tmp[ii] = pCUDA_Dg[l_incl[ii]];
          }

          for (ii = 0; ii < UNRL; ii++)
            tmp += l_dbr[ii] * l_tmp[ii];
        }
#pragma unroll 3
        for (; j < incl_count; j++)
        {
          int l_incl = incl[j];
          double l_dbr = dbr[j];

          tmp += l_dbr * pCUDA_Dg[l_incl];
        }
        __stwb(&dytempp[d], Scale * tmp);
      }
      pCUDA_Dg += mr;
      pCUDA_Dg1 += mr;
    }
  }
  else
  {
    double *__restrict__ p = dytempp + d;
#pragma unroll
    for (i = 1; i <= ncoef0 - (UNRL - 1); i += UNRL)
      for (int t = 0; t < UNRL; t++, p += Lpoints1)
        __stwb(p, 0.0);
#pragma unroll
    for (; i <= ncoef0; i++, p += Lpoints1)
      __stwb(p, 0.0);
  }

  return 0;
}