#include "hip/hip_runtime.h"
/* computes integrated brightness of all visible and iluminated areas
   and its derivatives

   8.11.2006
*/

#include <cmath>
#include "globals_CUDA.h"
#include <>

__device__ void matrix_neo(freq_context* CUDA_LCC, double cg[], int lnp1, int Lpoints)
{
    double f, cf, sf, pom, pom0, alpha;
    double ee_1, ee_2, ee_3, ee0_1, ee0_2, ee0_3, t, tmat;
    int lnp, jp;

    int brtmph, brtmpl, index;
    brtmph = Lpoints / CUDA_BLOCK_DIM;
    if (Lpoints % CUDA_BLOCK_DIM) brtmph++;
    brtmpl = threadIdx.x * brtmph;
    brtmph = brtmpl + brtmph;
    if (brtmph > Lpoints) brtmph = Lpoints;
    brtmpl++;
    double inv_coef = 1.0 / cg[CUDA_ncoef0 + 2];


    lnp = lnp1 + brtmpl - 1;
    for (jp = brtmpl; jp <= brtmph; jp++)
    {
        lnp++;
        ee_1 = CUDA_ee[lnp * 3 + 0];// position vectors
        ee0_1 = CUDA_ee0[lnp * 3 + 0];
        ee_2 = CUDA_ee[lnp * 3 + 1];
        ee0_2 = CUDA_ee0[lnp * 3 + 1];
        ee_3 = CUDA_ee[lnp * 3 + 2];
        ee0_3 = CUDA_ee0[lnp * 3 + 2];
        t = CUDA_tim[lnp];

        alpha = acos(ee_1 * ee0_1 + ee_2 * ee0_2 + ee_3 * ee0_3);

        /* Exp-lin model (const.term=1.) */
		f = exp(-alpha * inv_coef);
        (*CUDA_LCC).jp_Scale[jp] = __fma_rn(cg[CUDA_ncoef0 + 3], alpha, __fma_rn(cg[CUDA_ncoef0 + 1], f, 1));
		(*CUDA_LCC).jp_dphp_1[jp] = f;
		(*CUDA_LCC).jp_dphp_2[jp] = cg[CUDA_ncoef0 + 1] * f * alpha / (cg[CUDA_ncoef0 + 2] * cg[CUDA_ncoef0 + 2]);
		(*CUDA_LCC).jp_dphp_3[jp] = alpha;

        //  matrix start
        double f = __fma_rn(cg[CUDA_ncoef0], t, CUDA_Phi_0);
		f = fmod(f, 2 * PI);
        sincos(f, &sf, &cf);

        /* rotation matrix, Z axis, angle f */
        tmat = __fma_rn(cf, (*CUDA_LCC).Blmat[1][1], sf * (*CUDA_LCC).Blmat[2][1]);
		pom = tmat * ee_1;
		pom0 = tmat * ee0_1;
		tmat = __fma_rn(cf, (*CUDA_LCC).Blmat[1][2], sf * (*CUDA_LCC).Blmat[2][2]);
		pom += tmat * ee_2;
		pom0 += tmat * ee0_2;
		tmat = __fma_rn(cf, (*CUDA_LCC).Blmat[1][3], sf * (*CUDA_LCC).Blmat[2][3]);
		(*CUDA_LCC).e_1[jp] = __fma_rn(tmat, ee_3, pom);
		(*CUDA_LCC).e0_1[jp] = __fma_rn(tmat, ee0_3, pom0);

        tmat = __fma_rn(-sf, (*CUDA_LCC).Blmat[1][1], cf * (*CUDA_LCC).Blmat[2][1]);
		pom = tmat * ee_1;
		pom0 = tmat * ee0_1;
		tmat = __fma_rn(-sf, (*CUDA_LCC).Blmat[1][2], cf * (*CUDA_LCC).Blmat[2][2]);
		pom += tmat * ee_2;
		pom0 += tmat * ee0_2;
		tmat = __fma_rn(-sf, (*CUDA_LCC).Blmat[1][3], cf * (*CUDA_LCC).Blmat[2][3]);
		(*CUDA_LCC).e_2[jp] = __fma_rn(tmat, ee_3, pom);
		(*CUDA_LCC).e0_2[jp] = __fma_rn(tmat, ee0_3, pom0);

		tmat = (*CUDA_LCC).Blmat[3][1];
		pom = tmat * ee_1;
		pom0 = tmat * ee0_1;
		tmat = (*CUDA_LCC).Blmat[3][2];
		pom += tmat * ee_2;
		pom0 += tmat * ee0_2;
		tmat = (*CUDA_LCC).Blmat[3][3];
		(*CUDA_LCC).e_3[jp] = __fma_rn(tmat, ee_3, pom);
		(*CUDA_LCC).e0_3[jp] = __fma_rn(tmat, ee0_3, pom0);

		tmat = __fma_rn(cf, (*CUDA_LCC).Dblm[1][1][1], sf * (*CUDA_LCC).Dblm[1][2][1]);
		pom = tmat * ee_1;
		pom0 = tmat * ee0_1;
		tmat = __fma_rn(cf, (*CUDA_LCC).Dblm[1][1][2], sf * (*CUDA_LCC).Dblm[1][2][2]);
		pom += tmat * ee_2;
		pom0 += tmat * ee0_2;
		tmat = __fma_rn(cf, (*CUDA_LCC).Dblm[1][1][3], sf * (*CUDA_LCC).Dblm[1][2][3]);
        index = jp * 16 + 1 * 4 + 1;
        (*CUDA_LCC).de[index] = __fma_rn(tmat, ee_3, pom);
        (*CUDA_LCC).de0[index] = __fma_rn(tmat, ee0_3, pom0);

		tmat = __fma_rn(cf, (*CUDA_LCC).Dblm[2][1][1], sf * (*CUDA_LCC).Dblm[2][2][1]);
		pom = tmat * ee_1;
		pom0 = tmat * ee0_1;
		tmat = __fma_rn(cf, (*CUDA_LCC).Dblm[2][1][2], sf * (*CUDA_LCC).Dblm[2][2][2]);
		pom += tmat * ee_2;
		pom0 += tmat * ee0_2;
		tmat = __fma_rn(cf, (*CUDA_LCC).Dblm[2][1][3], sf * (*CUDA_LCC).Dblm[2][2][3]);
        index++; // index = jp * 16 + 1 * 4 + 2;
        (*CUDA_LCC).de[index] = __fma_rn(tmat, ee_3, pom);
        (*CUDA_LCC).de0[index] = __fma_rn(tmat, ee0_3, pom0);

		tmat = __fma_rn(-t * sf, (*CUDA_LCC).Blmat[1][1], t * cf * (*CUDA_LCC).Blmat[2][1]);
		pom = tmat * ee_1;
		pom0 = tmat * ee0_1;
		tmat = __fma_rn(-t * sf, (*CUDA_LCC).Blmat[1][2], t * cf * (*CUDA_LCC).Blmat[2][2]);
		pom += tmat * ee_2;
		pom0 += tmat * ee0_2;
		tmat = __fma_rn(-t * sf, (*CUDA_LCC).Blmat[1][3], t * cf * (*CUDA_LCC).Blmat[2][3]);
        index++; // index = jp * 16 + 1 * 4 + 3;
        (*CUDA_LCC).de[index] = __fma_rn(tmat, ee_3, pom);
        (*CUDA_LCC).de0[index] = __fma_rn(tmat, ee0_3, pom0);

		tmat = __fma_rn(-sf, (*CUDA_LCC).Dblm[1][1][1], cf * (*CUDA_LCC).Dblm[1][2][1]);
		pom = tmat * ee_1;
		pom0 = tmat * ee0_1;
		tmat = __fma_rn(-sf, (*CUDA_LCC).Dblm[1][1][2], cf * (*CUDA_LCC).Dblm[1][2][2]);
		pom += tmat * ee_2;
		pom0 += tmat * ee0_2;
		tmat = __fma_rn(-sf, (*CUDA_LCC).Dblm[1][1][3], cf * (*CUDA_LCC).Dblm[1][2][3]);
        index = jp * 16 + 2 * 4 + 1;
        (*CUDA_LCC).de[index] = __fma_rn(tmat, ee_3, pom);
        (*CUDA_LCC).de0[index] = __fma_rn(tmat, ee0_3, pom0);

		tmat = __fma_rn(-sf, (*CUDA_LCC).Dblm[2][1][1], cf * (*CUDA_LCC).Dblm[2][2][1]);
		pom = tmat * ee_1;
		pom0 = tmat * ee0_1;
		tmat = __fma_rn(-sf, (*CUDA_LCC).Dblm[2][1][2], cf * (*CUDA_LCC).Dblm[2][2][2]);
		pom += tmat * ee_2;
		pom0 += tmat * ee0_2;
		tmat = __fma_rn(-sf, (*CUDA_LCC).Dblm[2][1][3], cf * (*CUDA_LCC).Dblm[2][2][3]);
        index++; // index = jp * 16 + 2 * 4 + 2;
        (*CUDA_LCC).de[index] = __fma_rn(tmat, ee_3, pom);
        (*CUDA_LCC).de0[index] = __fma_rn(tmat, ee0_3, pom0);

		tmat = __fma_rn(-t * cf, (*CUDA_LCC).Blmat[1][1], (-t * sf) * (*CUDA_LCC).Blmat[2][1]);
		pom = tmat * ee_1;
		pom0 = tmat * ee0_1;
		tmat = __fma_rn(-t * cf, (*CUDA_LCC).Blmat[1][2], (-t * sf) * (*CUDA_LCC).Blmat[2][2]);
		pom += tmat * ee_2;
		pom0 += tmat * ee0_2;
		tmat = __fma_rn(-t * cf, (*CUDA_LCC).Blmat[1][3], (-t * sf) * (*CUDA_LCC).Blmat[2][3]);
        index++; // index = jp * 16 + 2 * 4 + 3;
        (*CUDA_LCC).de[index] = __fma_rn(tmat, ee_3, pom);
        (*CUDA_LCC).de0[index] = __fma_rn(tmat, ee0_3, pom0);

		tmat = (*CUDA_LCC).Dblm[1][3][1];
		pom = tmat * ee_1;
		pom0 = tmat * ee0_1;
		tmat = (*CUDA_LCC).Dblm[1][3][2];
		pom += tmat * ee_2;
		pom0 += tmat * ee0_2;
		tmat = (*CUDA_LCC).Dblm[1][3][3];
        index = jp * 16 + 3 * 4 + 1;
        (*CUDA_LCC).de[index] = __fma_rn(tmat, ee_3, pom);
        (*CUDA_LCC).de0[index] = __fma_rn(tmat, ee0_3, pom0);

		tmat = (*CUDA_LCC).Dblm[2][3][1];
		pom = tmat * ee_1;
		pom0 = tmat * ee0_1;
		tmat = (*CUDA_LCC).Dblm[2][3][2];
		pom += tmat * ee_2;
		pom0 += tmat * ee0_2;
		tmat = (*CUDA_LCC).Dblm[2][3][3];
        index++; // index = jp * 16 + 3 * 4 + 2;
        (*CUDA_LCC).de[index] = __fma_rn(tmat, ee_3, pom);
        (*CUDA_LCC).de0[index] = __fma_rn(tmat, ee0_3, pom0);

        index++; // index = jp * 16 + 3 * 4 + 3;
        (*CUDA_LCC).de[index] = 0;
        (*CUDA_LCC).de0[index] = 0;
    }
    __syncthreads();
}
__device__ double bright(freq_context* CUDA_LCC, double cg[], int jp, int Lpoints1, int Inrel)
{
    int ncoef0, ncoef, i, j, incl_count = 0;
    double cl, cls, dnom, s, Scale;
    double e_1, e_2, e_3, e0_1, e0_2, e0_3, de[4][4], de0[4][4];

    ncoef0 = CUDA_ncoef0;//ncoef - 2 - CUDA_Nphpar;
    ncoef = CUDA_ma;
    cl = exp(cg[ncoef - 1]); /* Lambert */
    cls = cg[ncoef];       /* Lommel-Seeliger */

    /* matrix from neo */
    /* derivatives */

    e_1 = (*CUDA_LCC).e_1[jp];
    e_2 = (*CUDA_LCC).e_2[jp];
    e_3 = (*CUDA_LCC).e_3[jp];
    e0_1 = (*CUDA_LCC).e0_1[jp];
    e0_2 = (*CUDA_LCC).e0_2[jp];
    e0_3 = (*CUDA_LCC).e0_3[jp];

    // Loop over the indices to map to flattened array
    for (int i = 1; i <= 3; ++i) 
    {
        for (int j = 1; j <= 3; ++j) 
        {
            // Calculate the flattened index for 'de' and 'de0'
            int index = jp * (4 * 4) + i * 4 + j;
            de[i][j] = (*CUDA_LCC).de[index];
            de0[i][j] = (*CUDA_LCC).de0[index];
        }
    }

    // index = x * 16 + y * 4 + z;

    /* Directions (and ders.) in the rotating system */

    //
    /*Integrated brightness (phase coeff. used later) */
    double lmu, lmu0, dsmu, dsmu0, sum1, sum10, sum2, sum20, sum3, sum30;
    double br, ar, tmp1, tmp2, tmp3, tmp4, tmp5;
    //   short int *incl=&(*CUDA_LCC).incl[threadIdx.x*MAX_N_FAC];
    //   double *dbr=&(*CUDA_LCC).dbr[threadIdx.x*MAX_N_FAC];
    short int incl[MAX_N_FAC];
    double dbr[MAX_N_FAC];
    //int2 bfr;

    br = 0;
    tmp1 = 0;
    tmp2 = 0;
    tmp3 = 0;
    tmp4 = 0;
    tmp5 = 0;
    j = blockIdx.x * (CUDA_Numfac1)+1;
    for (i = 1; i <= CUDA_Numfac; i++, j++)
    {
        lmu = __fma_rn(e_1, CUDA_Nor[i][0], __fma_rn(e_2, CUDA_Nor[i][1], e_3 * CUDA_Nor[i][2]));
		lmu0 = __fma_rn(e0_1, CUDA_Nor[i][0], __fma_rn(e0_2, CUDA_Nor[i][1], e0_3 * CUDA_Nor[i][2]));
        if ((lmu > TINY) && (lmu0 > TINY))
        {
            dnom = lmu + lmu0;
            s = lmu * lmu0 * (cl + cls / dnom);
            ar = CUDA_Area[j];
            br += ar * s;
            //
            incl[incl_count] = i;
            dbr[incl_count] = CUDA_Darea[i] * s;
            incl_count++;

            double lmu0_dnom = lmu0 / dnom;
            dsmu = __fma_rn(cls, lmu0_dnom * lmu0_dnom, cl * lmu0);
            double lmu_dnom = lmu / dnom;
            dsmu0 = __fma_rn(cls, lmu_dnom * lmu_dnom, cl * lmu);

            sum1 = __fma_rn(CUDA_Nor[i][0], de[1][1], __fma_rn(CUDA_Nor[i][1], de[2][1], (CUDA_Nor[i][2] * de[3][1])));
			sum10 = __fma_rn(CUDA_Nor[i][0], de0[1][1], __fma_rn(CUDA_Nor[i][1], de0[2][1], (CUDA_Nor[i][2] * de0[3][1])));
			tmp1 += ar * __fma_rn(dsmu, sum1, dsmu0 * sum10);
			sum2 = __fma_rn(CUDA_Nor[i][0], de[1][2], __fma_rn(CUDA_Nor[i][1], de[2][2], (CUDA_Nor[i][2] * de[3][2])));
			sum20 = __fma_rn(CUDA_Nor[i][0], de0[1][2], __fma_rn(CUDA_Nor[i][1], de0[2][2], (CUDA_Nor[i][2] * de0[3][2])));
			tmp2 += ar * __fma_rn(dsmu, sum2, dsmu0 * sum20);
			sum3 = __fma_rn(CUDA_Nor[i][0], de[1][3], __fma_rn(CUDA_Nor[i][1], de[2][3], (CUDA_Nor[i][2] * de[3][3])));
			sum30 = __fma_rn(CUDA_Nor[i][0], de0[1][3], __fma_rn(CUDA_Nor[i][1], de0[2][3], (CUDA_Nor[i][2] * de0[3][3])));
			tmp3 += ar * __fma_rn(dsmu, sum3, dsmu0 * sum30);

			tmp4 = __fma_rn(lmu * lmu0, ar, tmp4);
			double inv_sum = 1.0 / (lmu + lmu0);
			tmp5 = __fma_rn(ar * lmu * lmu0, inv_sum, tmp5);
        }
    }
    Scale = (*CUDA_LCC).jp_Scale[jp];
    i = jp + (ncoef0 - 3 + 1) * Lpoints1;

    /* Ders. of brightness w.r.t. rotation parameters */
    (*CUDA_LCC).dytemp[i] = Scale * tmp1;
    i += Lpoints1;
    (*CUDA_LCC).dytemp[i] = Scale * tmp2;
    i += Lpoints1;
    (*CUDA_LCC).dytemp[i] = Scale * tmp3;
    i += Lpoints1;

    /* Ders. of br. w.r.t. phase function params. */
    (*CUDA_LCC).dytemp[i] = br * (*CUDA_LCC).jp_dphp_1[jp];
    i += Lpoints1;
    (*CUDA_LCC).dytemp[i] = br * (*CUDA_LCC).jp_dphp_2[jp];
    i += Lpoints1;
    (*CUDA_LCC).dytemp[i] = br * (*CUDA_LCC).jp_dphp_3[jp];

    /* Ders. of br. w.r.t. cl, cls */
    (*CUDA_LCC).dytemp[jp + (ncoef - 1) * (Lpoints1)] = Scale * tmp4 * cl;
    (*CUDA_LCC).dytemp[jp + (ncoef) * (Lpoints1)] = Scale * tmp5;

    /* Scaled brightness */
    (*CUDA_LCC).ytemp[jp] = br * Scale;

    ncoef0 -= 3;
    int m, m1, mr, iStart;
    int d, d1, dr;

    iStart = Inrel + 1;
    m = blockIdx.x * CUDA_Dg_block + iStart * (CUDA_Numfac1);
    d = jp + (Lpoints1 << Inrel);

    m1 = m + (CUDA_Numfac1);
    mr = 2 * CUDA_Numfac1;
    d1 = d + (Lpoints1);
    dr = 2 * Lpoints1;
    /* Derivatives of brightness w.r.t. g-coeffs */
    if (incl_count)
    {
        for (i = iStart; i <= ncoef0; i += 2, m += mr, m1 += mr, d += dr, d1 += dr)
        {
            double tmp = 0, tmp1 = 0;

            double l_dbr = dbr[0];
            int l_incl = incl[0];
            int is_next_coef_valid = (i + 1) <= ncoef0;

            tmp = l_dbr * CUDA_Dg[m + l_incl];
            if (is_next_coef_valid)
            {
                tmp1 = l_dbr * CUDA_Dg[m1 + l_incl];
            }

            for (j = 1; j < incl_count; j++)
            {
                double l_dbr = dbr[j];
                int l_incl = incl[j];
                tmp += l_dbr * CUDA_Dg[m + l_incl];
                if (is_next_coef_valid)
                {
                    tmp1 += l_dbr * CUDA_Dg[m1 + l_incl];
                }
            }

            (*CUDA_LCC).dytemp[d] = Scale * tmp;
            if (is_next_coef_valid)
            {
                (*CUDA_LCC).dytemp[d1] = Scale * tmp1;
            }
        }
    }
    else
    {
        for (i = 1; i <= ncoef0; i++, d += Lpoints1)
            (*CUDA_LCC).dytemp[d] = 0;
    }

    return(0);
}
