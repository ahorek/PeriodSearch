#include "hip/hip_runtime.h"
/* beta, lambda rotation matrix and its derivatives

   8.11.2006
*/

#include <math.h>
#include "globals_CUDA.h"

__device__ void __forceinline__ blmatrix(double bet, double lam, int tid)
{
   double cb, sb, cl, sl, cbcl, cbsl, sbcl, sbsl, nsb, ncb, nsl, ncl;
   //__builtin_assume(bet > (-2.0 * PI) && bet < (2.0 * PI));
   sincos(bet, &sb, &cb);

   //__builtin_assume(lam > (-2.0 * PI) && lam < (2.0 * PI));
   sincos(lam, &sl, &cl);

   nsb = -sb;
   ncb = -cb;
   cbcl = cb * cl;
   cbsl = cb * sl;
   nsl = -sl;

   Blmat[1][1][tid] = cl;
   Blmat[2][2][tid] = cb;
   Blmat[0][0][tid] = cbcl;
   Dblm[0][2][0][tid] = cbcl;
   Dblm[1][0][1][tid] = cbcl;
   Blmat[0][1][tid] = cbsl;
   Dblm[0][2][1][tid] = cbsl;
   Dblm[1][0][0][tid] = -cbsl;
   Blmat[0][2][tid] = nsb;
   Dblm[0][2][2][tid] = nsb;
   Blmat[1][0][tid] = nsl;
   Dblm[1][1][1][tid] = nsl;
   Blmat[1][2][tid] = 0;

   sbcl = sb * cl;
   sbsl = sb * sl;
   ncl = -cl;
   double nsbcl = -sbcl;
   double nsbsl = -sbsl;

   Blmat[2][0][tid] = sbcl;
   Dblm[1][2][1][tid] = sbcl;
   Dblm[0][0][0][tid] = nsbcl;
   Blmat[2][1][tid] = sbsl;
   Dblm[0][0][1][tid] = nsbsl;
   Dblm[1][2][0][tid] = nsbsl;
   Dblm[1][1][0][tid] = ncl;
   Dblm[0][0][2][tid] = ncb;

   // Ders. of Blmat w.r.t. bet
   Dblm[0][1][0][tid] = 0;
   Dblm[0][1][1][tid] = 0;
   Dblm[0][1][2][tid] = 0;

   // Ders. w.r.t. lam
   Dblm[1][0][2][tid] = 0;
   Dblm[1][1][2][tid] = 0;
   Dblm[1][2][2][tid] = 0;
}
