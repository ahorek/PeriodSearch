#include "hip/hip_runtime.h"
//#ifndef __HIPCC__
//#define __HIPCC__
//#endif

#include <stdio.h>
#include <stdlib.h>
#include "globals_CUDA.h"
#include "declarations_CUDA.h"
//#include <hip/hip_runtime.h>
#include <>

__device__ void __forceinline__ MrqcofCurve23I0IA0(freq_context * __restrict__ CUDA_LCC, double * __restrict__ alpha, double * __restrict__ beta, int bid)
{
  int lpoints = 3;
  int mf1 = CUDA_mfit1;
  int l, jp, j, k, m, lnp2, Lpoints1 = lpoints + 1;
  double dy, sig2i, wt, ymod, wght, ltrial_chisq;
  __shared__ double dyda[BLOCKX4][N80];
  double * __restrict__ dydap = dyda[threadIdx.y];
  //__syncthreads();

  if(threadIdx.x == 0)
    {
      npg1[bid] += lpoints;
    }

  lnp2 = npg2[bid];
  ltrial_chisq = trial_chisqg[bid];

  int ma = CUDA_ma, lma = CUDA_lastma;
  int lastone = CUDA_lastone;
  int * __restrict__ iapp = CUDA_ia;
  double * __restrict__ dytemp = CUDA_LCC->dytemp, * __restrict__ ytemp = CUDA_LCC->ytemp;;
  
#pragma unroll 
  for(jp = 1; jp <= lpoints; jp++)
    {
      int ixx = jp + (threadIdx.x + 1) * Lpoints1; // ZZZ bad, strided read, BAD
      double * __restrict__ c = &(dytemp[ixx]);
      l = threadIdx.x;
#pragma unroll 2
      while(l < ma)
	{
	  dydap[l] = __ldca(c); // YYYY
	  l += CUDA_BLOCK_DIM;
	  c += CUDA_BLOCK_DIM * Lpoints1;
	}
      
      __syncwarp();
      
      lnp2++;
      double s = __ldg(&CUDA_sig[lnp2]);
      ymod = __ldca(&ytemp[jp]);
      sig2i = ___drcp_rn(s * s);
      wght = __ldg(&CUDA_Weight[lnp2]);
      dy = __ldg(&CUDA_brightness[lnp2]) - ymod;
      
      j = 0;
      double sig2iwght = sig2i * wght;
      
#pragma unroll 
      for(l = 2; l <= lastone; l++)
	{
	  j++;
	  wt = dydap[l-1] * sig2iwght;
	  
	  int xx = threadIdx.x + 1;
	  double * __restrict__ alp = &alpha[j * mf1 + xx - 1];
#pragma unroll 2
	  while(xx <= l)
	    {
	      //if(xx != 0)
	      double const * __restrict__ alp2 = alp;
	      __stwb(alp, *alp2 + wt * dydap[xx-1]);
	      xx  += CUDA_BLOCK_DIM;
	      alp += CUDA_BLOCK_DIM;
	    } /* m */
	  //__syncthreads();
	  if(threadIdx.x == 0)
	    {
	      beta[j] = beta[j] + dy * wt;
	    }
	  //__syncthreads();
	} /* l */
      
#pragma unroll 
      for(; l <= lma; l++)
	{
	  if(iapp[l])
	    {
	      j++;
	      wt = dydap[l-1] * sig2iwght;
	      
	      int xx = threadIdx.x + 1;
	      double * __restrict__ alph = &alpha[j * mf1 - 1];
#pragma unroll 2
	      while(xx <= lastone)
		{
		  //if(xx != 0)
		  __stwb(&alph[xx], alph[xx] + wt * dydap[xx-1]);
		  xx += CUDA_BLOCK_DIM;
		} /* m */
	      //__syncthreads();
	      if(threadIdx.x == 0)
		{
		  k = lastone - 1;
		  m = lastone + 1;
		  int * __restrict__ iap = iapp + m;
		  double * __restrict__ alp = alpha + j * mf1 + k;
#pragma unroll 4
		  for(; m <= l; m++)
		    {
		      if(*iap)
			{
			  alp++;
			  __stwb(alp, *alp + wt * dydap[m-1]);
			}
		      iap++;
		    } /* m */
		  beta[j] = beta[j] + dy * wt;
		}
	      //__syncthreads();
	    }
	} /* l */
      ltrial_chisq = ltrial_chisq + dy * dy * sig2iwght;
    } /* jp */

  if(threadIdx.x == 0)
    {
      npg2[bid] = lnp2;
      trial_chisqg[bid] = ltrial_chisq;
    }
}

__device__ void __forceinline__ MrqcofCurve23I0IA1(freq_context * __restrict__ CUDA_LCC, double * __restrict__ alpha, double * __restrict__ beta, int bid)
{
  int lpoints = 3;
  int mf1 = CUDA_mfit1;
  int l, jp, j, k, m, lnp2, Lpoints1 = lpoints + 1;
  double dy, sig2i, wt, ymod, wght, ltrial_chisq;
  __shared__ double dyda[N80];
  
  __syncwarp();

  if(threadIdx.x == 0)
    {
      npg1[bid] += lpoints;
    }

  lnp2 = npg2[bid];
  ltrial_chisq = trial_chisqg[bid];

  int ma = CUDA_ma, lma = CUDA_lastma;
  int lastone = CUDA_lastone;
  int * __restrict__ iapp = CUDA_ia;
  double * __restrict__ dytemp = CUDA_LCC->dytemp, * __restrict__ ytemp = CUDA_LCC->ytemp;
#pragma unroll 
  for(jp = 1; jp <= lpoints; jp++) 
    {
      lnp2++;
      double s = __ldg(&CUDA_sig[lnp2]);
      ymod = __ldca(&(ytemp[jp]));
      sig2i = ___drcp_rn(s * s); 
      wght = __ldg(&CUDA_Weight[lnp2]);
      dy = __ldg(&CUDA_brightness[lnp2]) - ymod;
      
      int ixx = jp + (threadIdx.x + 1) * Lpoints1; // ZZZ, bad, strided read, BAD!
      double * __restrict__ c = &(dytemp[ixx]); //  bad c
      l = threadIdx.x + 1;
#pragma unroll 4
      while(l <= ma - CUDA_BLOCK_DIM)
	{
	  double a, b;
	  a = __ldca(c);
	  c += CUDA_BLOCK_DIM * Lpoints1;
	  b = __ldca(c);
	  c += CUDA_BLOCK_DIM * Lpoints1;
	  dyda[l-1] = a;
	  dyda[l-1 + CUDA_BLOCK_DIM] = b;
	  l += 2*CUDA_BLOCK_DIM;
	}
#pragma unroll 1
      while(l <= ma)
	{
	  dyda[l-1] = __ldca(c);
	  l += CUDA_BLOCK_DIM;
	  c += CUDA_BLOCK_DIM * Lpoints1;
	}
      
      __syncwarp();
      
      j = 0;
      double sig2iwght = sig2i * wght;
      
#pragma unroll 4
      for(l = 1; l <= lastone; l++)
	{
	  j++;
	  wt = dyda[l-1] * sig2iwght;
	  int xx = threadIdx.x + 1;
#pragma unroll 2
	  while(xx <= l)
	    {
	      alpha[j * mf1 + xx] += wt * dyda[xx-1];
	      xx += CUDA_BLOCK_DIM;
	    } /* m */
	  //__syncthreads();
	  if(threadIdx.x == 0)
	    {
	      beta[j] = beta[j] + dy * wt;
	    }
	  //__syncthreads();
	} /* l */
      
#pragma unroll 4
      for(; l <= lma; l++)
	{
	  if(iapp[l])
	    {
	      j++;
	      wt = dyda[l-1] * sig2iwght;
	      int xx = threadIdx.x + 1;
#pragma unroll 2
	      while(xx <= lastone)
		{
		  //if(xx != 0)
		  alpha[j * mf1 + xx] += wt * dyda[xx-1];
		  xx += CUDA_BLOCK_DIM;
		} /* m */
	      //__syncthreads();
	      if(threadIdx.x == 0)
		{
		  k = lastone;
		  m = lastone + 1;
		  int * __restrict__ iap = iapp + m;
		  double * __restrict__ alp = alpha + j * mf1 + k;
#pragma unroll 4
		  for(; m <= l; m++)
		    {
		      if(*iap)
			{
			  alp++;
			  __stwb(alp, *alp + wt * dyda[m-1]);
			}
		      iap++;
		    } /* m */
		  beta[j] = beta[j] + dy * wt;
		}
	      //__syncthreads();
	    }
	} /* l */
      ltrial_chisq = ltrial_chisq + dy * dy * sig2iwght;
    } /* jp */

  if(threadIdx.x == 0)
    {
      npg2[bid] = lnp2;
      trial_chisqg[bid] = ltrial_chisq;
    }
}

// SLOW (only 3 threads participate -> 1/10 perf))
  __device__ void __forceinline__ MrqcofCurve23I1IA0(freq_context * __restrict__ CUDA_LCC, double * __restrict__ alpha, double * __restrict__ beta, int bid)
{
  int lpoints = 3;
  int mf1 = CUDA_mfit1;
  int l, jp, j, k, m, lnp1, lnp2, Lpoints1 = lpoints + 1;
  double dy, sig2i, wt, ymod, coef1, coef, wght, ltrial_chisq;
  __shared__ double dydat[3][N80];
  
  lnp1 = npg1[bid] + threadIdx.x + 1;

  int ma = CUDA_ma;
  //int bid = blockIdx();
  jp = threadIdx.x + 1;
  double rave = ___drcp_rn(aveg[bid]);
  double * __restrict__ dytmpp = CUDA_LCC->dytemp, * __restrict__ cuda_sig = CUDA_sig, * __restrict__ ytemp = CUDA_LCC->ytemp;
  double * __restrict__ cuda_weight = CUDA_Weight, * __restrict__ cuda_brightness = CUDA_brightness;
  //double * __restrict__ dave = CUDA_LCC->dave;
  double * __restrict__ davep = &(dave[bid][0]);
  long int lpadd = sizeof(double) * Lpoints1;
  
  //#pragma unroll 
  if(jp <= lpoints)
    {
      int ixx = jp + Lpoints1;
      // Set the size scale coeff. deriv. explicitly zero for relative lcurves 
      dytmpp[ixx] = 0; // YYY, good, consecutive
      coef = cuda_sig[lnp1] * lpoints * rave; // / CUDA_LCC->ave;
      
      double yytmp = ytemp[jp];
      coef1 = yytmp * rave; // / CUDA_LCC->ave;
      ytemp[jp] = coef * yytmp;
      
      ixx += Lpoints1;
      double * __restrict__ dyp = dytmpp + ixx; //&(CUDA_LCC->dytemp[ixx]);
      double * __restrict__ dap = &(davep[2]);
#pragma unroll 
      for(l = 2; l <= ma - (UNRL - 1); l += UNRL, ixx += UNRL * Lpoints1)
	{
	  double dd[UNRL], dy[UNRL];
	  int ii;
	  double * __restrict__ dypp = dyp;
	  for(ii = 0; ii < UNRL; ii++)
	    {
	      dy[ii] = __ldg(dypp);
	      //dypp += Lpoints1;
	      dypp = (double *)(((char *)dypp) + lpadd);

	      dd[ii] = __ldca(dap);
	      dap++;
	    }
	  for(ii = 0; ii < UNRL; ii++)
	    {
	      __stwb(dyp, coef * (dy[ii] - coef1 * dd[ii])); //WXX
	      //dyp += Lpoints1;
	      dyp = (double *)(((char *)dyp) + lpadd);
	    }
	}
#pragma unroll 
      for(; l <= ma; l++, dyp += Lpoints1, dap++)
	__stwb(dyp, coef * ( __ldg(dyp) - coef1 * __ldca(dap))); //WXX
	//*dyp = __ldg(dyp) * coef - coef1 * __ldg(dap);

      jp += CUDA_BLOCK_DIM;
      lnp1 += CUDA_BLOCK_DIM;
    }

  __syncwarp();

  if(threadIdx.x == 0)
    {
      npg1[bid] += lpoints;
    }

  lnp2 = npg2[bid];
  ltrial_chisq = trial_chisqg[bid];
  int lastone = CUDA_lastone;
  int * __restrict__ iapp = CUDA_ia;

#pragma unroll 
  for(jp = 1; jp <= lpoints; jp++)
    {
      if(jp == 1)
	{
	  int ixx = jp + (threadIdx.x + 1) * Lpoints1; // RXX bad, strided read, BAD
	  double * __restrict__ c = dytmpp + ixx;  //&(CUDA_LCC->dytemp[ixx]);
	  l = threadIdx.x;
#pragma unroll 2
	  while(l < ma)
	    {
	      dydat[0][l] = c[0]; // YYYY RXX
	      dydat[1][l] = c[1]; // YYYY
	      dydat[2][l] = c[2]; // YYYY
	      l += CUDA_BLOCK_DIM;
	      c += CUDA_BLOCK_DIM * Lpoints1;
	    }
	  __syncwarp();
	}
      
      double * __restrict__ dyda = &dydat[jp-1][0];
      
      lnp2++;
      double s = cuda_sig[lnp2];
      ymod = ytemp[jp];
      sig2i = ___drcp_rn(s * s);
      wght = cuda_weight[lnp2];
      dy = cuda_brightness[lnp2] - ymod;
      
      j = 0;
      double sig2iwght = sig2i * wght;

      double * __restrict__ dydap = dyda + 1;
#pragma unroll 
      for(l = 2; l <= lastone; l++)
	{
	  j++;
	  wt = *dydap * sig2iwght;
	  dydap++;
	  
	  int xx = threadIdx.x + 1;
	  double * __restrict__ alp = &(alpha[j * mf1 - 1 + xx]);
#pragma unroll 2
	  while(xx <= l)
	    {
	      //if(xx != 0)
	      double * __restrict__ alp2 = alp;
	      __stwb(alp, *alp2 + wt * dyda[xx-1]);
	      xx += CUDA_BLOCK_DIM;
	      alp += CUDA_BLOCK_DIM;
	    } /* m */
	  //__syncthreads();
	  if(threadIdx.x == 0)
	    {
	      beta[j] += dy * wt;
	    }
	  //__syncthreads();
	} /* l */
      
#pragma unroll 
      for(; l <= CUDA_lastma; l++)
	{
	  if(iapp[l])
	    {
	      j++;
	      wt = *dydap * sig2iwght;
	      
	      int xx = threadIdx.x + 1;
	      double * __restrict__ alp = &alpha[j * mf1 - 1];
#pragma unroll 2
	      while(xx <= lastone)
		{
		  //if(xx != 0)
		  double const * __restrict__ alp2 = alp;
		  __stwb(alp, *alp2 + wt * dyda[xx-1]);
		  xx += CUDA_BLOCK_DIM;
		  alp += CUDA_BLOCK_DIM;
		} /* m */
	      //__syncthreads();
	      if(threadIdx.x == 0)
		{
		  k = lastone - 1;
		  m = lastone + 1;
		  int * __restrict__ iap = iapp + m;
		  double * __restrict__ alp = alpha + j * mf1 + k;
#pragma unroll 4
		  for(; m <= l; m++)
		    {
		      if(*iap)
			{
			  //k++;
			  alp++;
			  double const * __restrict__ alp2 = alp;
			  __stwb(alp, *alp2 + wt * dyda[m - 1]);
			}
		      iap++;
		    } /* m */
		  beta[j] = beta[j] + dy * wt;
		}
	      //__syncthreads();
	    }
	} /* l */
      ltrial_chisq = ltrial_chisq + dy * dy * sig2iwght;
    } /* jp */

  if(threadIdx.x == 0)
    {
      npg2[bid] = lnp2;
      trial_chisqg[bid] = ltrial_chisq;
    }
}

__global__ void
__launch_bounds__(768) 
CudaCalculateIter1Mrqcof1Curve2I0IA0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I0IA0(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}


__global__ void
__launch_bounds__(768) 
CudaCalculateIter1Mrqcof1Curve2I0IA1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I0IA1(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}


__global__ void
__launch_bounds__(768) 
CudaCalculateIter1Mrqcof1Curve2I1IA0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I1IA0(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}

__global__ void
__launch_bounds__(768) 
CudaCalculateIter1Mrqcof1Curve2I1IA1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I1IA1(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}

__global__ 
void CudaCalculateIter1Mrqcof2CurveM12I0IA1(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  double *atryp = atry[bid]; //CUDA_LCC->atry;
  mrqcof_matrix(CUDA_LCC, atryp, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, atryp, 0, lpoints, bid);
  MrqcofCurve2I0IA1(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, lpoints, bid);
}

__global__ 
void CudaCalculateIter1Mrqcof2CurveM12I0IA0(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  double *atryp = atry[bid]; //CUDA_LCC->atry;
  mrqcof_matrix(CUDA_LCC, atryp, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, atryp, 0, lpoints, bid);
  MrqcofCurve2I0IA0(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, lpoints, bid);
}

__global__ void CudaCalculateIter1Mrqcof2Curve2I0IA0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  MrqcofCurve23I0IA0(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, bid);
}

__global__ void CudaCalculateIter1Mrqcof2Curve2I0IA1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  MrqcofCurve23I0IA1(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, bid);
}


// SLOW
__global__ void CudaCalculateIter1Mrqcof2Curve2I1IA0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  MrqcofCurve23I1IA0(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, bid);
}



__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof2Curve2I1IA1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I1IA1(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, bid);
}



__global__ 
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1CurveM12I0IA0(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  double *cg = CUDA_LCC->cg;
  mrqcof_matrix(CUDA_LCC, cg, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, cg, 0, lpoints, bid);
  MrqcofCurve2I0IA0(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, lpoints, bid);
}


__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1CurveM12I0IA1(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  double *cg = CUDA_LCC->cg;
  mrqcof_matrix(CUDA_LCC, cg, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, cg, 0, lpoints, bid);
  MrqcofCurve2I0IA1(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, lpoints, bid);
}



__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1CurveM12I1IA0(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  double *cg = CUDA_LCC->cg;
  mrqcof_matrix(CUDA_LCC, cg, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, cg, 1, lpoints, bid);
  MrqcofCurve2I1IA0(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, lpoints, bid);
}


__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1CurveM12I1IA1(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  double *cg = CUDA_LCC->cg;
  mrqcof_matrix(CUDA_LCC, cg, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, cg, 1, lpoints, bid);
  MrqcofCurve2I1IA1(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, lpoints, bid);
}


__global__ 
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1Curve1LastI0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  if(CUDA_LCC->ytemp == NULL) return;

  mrqcof_curve1_lastI0(CUDA_LCC, CUDA_LCC->cg, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}


__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1Curve1LastI1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  mrqcof_curve1_lastI1(CUDA_LCC, CUDA_LCC->cg, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}

__global__ 
void CudaCalculateIter1Mrqcof2CurveM12I1IA1(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  double *atryp = atry[bid]; //CUDA_LCC->atry;
  mrqcof_matrix(CUDA_LCC, atryp, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, atryp, 1, lpoints, bid);
  MrqcofCurve2I1IA1(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, lpoints, bid);
}

__global__ 
__launch_bounds__(384) 
void CudaCalculateIter1Mrqcof2CurveM12I1IA0(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  double *atryp = atry[bid]; //CUDA_LCC->atry;
  mrqcof_matrix(CUDA_LCC, atryp, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, atryp, 1, lpoints, bid);
  MrqcofCurve2I1IA0(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, lpoints, bid);
}

__global__
__launch_bounds__(768) 
void CudaCalculateIter1Mrqcof2Curve1LastI0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  mrqcof_curve1_lastI0(CUDA_LCC, atry[bid], CUDA_LCC->covar, CUDA_LCC->da, bid);
}

__global__
__launch_bounds__(1024) 
void CudaCalculateIter1Mrqcof2Curve1LastI1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  mrqcof_curve1_lastI1(CUDA_LCC, atry[bid], CUDA_LCC->covar, CUDA_LCC->da, bid);
}