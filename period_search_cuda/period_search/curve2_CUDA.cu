#include "hip/hip_runtime.h"
//#ifndef __HIPCC__
//#define __HIPCC__
//#endif

#include <stdio.h>
#include <stdlib.h>
#include "globals_CUDA.h"
#include "declarations_CUDA.h"
//#include <hip/hip_runtime.h>
#include <>

__device__ void __forceinline__ MrqcofCurve23I0IA0(freq_context * __restrict__ CUDA_LCC, double * __restrict__ alpha, double * __restrict__ beta, int bid)
{
  int lpoints = 3;
  int mf1 = CUDA_mfit1;
  int l, jp, j, k, m, lnp2, Lpoints1 = lpoints + 1;
  double dy, sig2i, wt, ymod, wght, ltrial_chisq;
  __shared__ double dyda[BLOCKX4][N80];
  double * __restrict__ dydap = dyda[threadIdx.y];
  //__syncthreads();

  if(threadIdx.x == 0)
    {
      npg1[bid] += lpoints;
    }

  lnp2 = npg2[bid];
  ltrial_chisq = trial_chisqg[bid];

  int ma = CUDA_ma, lma = CUDA_lastma;
  int lastone = CUDA_lastone;
  int * __restrict__ iapp = CUDA_ia;
  double * __restrict__ dytemp = CUDA_LCC->dytemp, * __restrict__ ytemp = CUDA_LCC->ytemp;;
  
#pragma unroll 
  for(jp = 1; jp <= lpoints; jp++)
    {
      int ixx = jp + (threadIdx.x + 1) * Lpoints1; // ZZZ bad, strided read, BAD
      double * __restrict__ c = &(dytemp[ixx]);
      l = threadIdx.x;
#pragma unroll 2
      while(l < ma)
	{
	  dydap[l] = __ldca(c); // YYYY
	  l += CUDA_BLOCK_DIM;
	  c += CUDA_BLOCK_DIM * Lpoints1;
	}
      
      __syncwarp();
      
      lnp2++;
      double s = __ldg(&CUDA_sig[lnp2]);
      ymod = __ldca(&ytemp[jp]);
      sig2i = ___drcp_rn(s * s);
      wght = __ldg(&CUDA_Weight[lnp2]);
      dy = __ldg(&CUDA_brightness[lnp2]) - ymod;
      
      j = 0;
      double sig2iwght = sig2i * wght;
      
#pragma unroll 
      for(l = 2; l <= lastone; l++)
	{
	  j++;
	  wt = dydap[l-1] * sig2iwght;
	  
	  int xx = threadIdx.x + 1;
	  double * __restrict__ alp = &alpha[j * mf1 + xx - 1];
#pragma unroll 2
	  while(xx <= l)
	    {
	      //if(xx != 0)
	      double const * __restrict__ alp2 = alp;
	      __stwb(alp, *alp2 + wt * dydap[xx-1]);
	      xx  += CUDA_BLOCK_DIM;
	      alp += CUDA_BLOCK_DIM;
	    } /* m */
	  //__syncthreads();
	  if(threadIdx.x == 0)
	    {
	      beta[j] = beta[j] + dy * wt;
	    }
	  //__syncthreads();
	} /* l */
      
#pragma unroll 
      for(; l <= lma; l++)
	{
	  if(iapp[l])
	    {
	      j++;
	      wt = dydap[l-1] * sig2iwght;
	      
	      int xx = threadIdx.x + 1;
	      double * __restrict__ alph = &alpha[j * mf1 - 1];
#pragma unroll 2
	      while(xx <= lastone)
		{
		  //if(xx != 0)
		  __stwb(&alph[xx], alph[xx] + wt * dydap[xx-1]);
		  xx += CUDA_BLOCK_DIM;
		} /* m */
	      //__syncthreads();
	      if(threadIdx.x == 0)
		{
		  k = lastone - 1;
		  m = lastone + 1;
		  int * __restrict__ iap = iapp + m;
		  double * __restrict__ alp = alpha + j * mf1 + k;
#pragma unroll 4
		  for(; m <= l; m++)
		    {
		      if(*iap)
			{
			  alp++;
			  __stwb(alp, *alp + wt * dydap[m-1]);
			}
		      iap++;
		    } /* m */
		  beta[j] = beta[j] + dy * wt;
		}
	      //__syncthreads();
	    }
	} /* l */
      ltrial_chisq = ltrial_chisq + dy * dy * sig2iwght;
    } /* jp */

  if(threadIdx.x == 0)
    {
      npg2[bid] = lnp2;
      trial_chisqg[bid] = ltrial_chisq;
    }
}

__device__ void __forceinline__ MrqcofCurve23I0IA1(freq_context * __restrict__ CUDA_LCC, double * __restrict__ alpha, double * __restrict__ beta, int bid)
{
  int lpoints = 3;
  int mf1 = CUDA_mfit1;
  int l, jp, j, k, m, lnp2, Lpoints1 = lpoints + 1;
  double dy, sig2i, wt, ymod, wght, ltrial_chisq;
  __shared__ double dyda[N80];
  
  __syncwarp();

  if(threadIdx.x == 0)
    {
      npg1[bid] += lpoints;
    }

  lnp2 = npg2[bid];
  ltrial_chisq = trial_chisqg[bid];

  int ma = CUDA_ma, lma = CUDA_lastma;
  int lastone = CUDA_lastone;
  int * __restrict__ iapp = CUDA_ia;
  double * __restrict__ dytemp = CUDA_LCC->dytemp, * __restrict__ ytemp = CUDA_LCC->ytemp;
#pragma unroll 
  for(jp = 1; jp <= lpoints; jp++) 
    {
      lnp2++;
      double s = __ldg(&CUDA_sig[lnp2]);
      ymod = __ldca(&(ytemp[jp]));
      sig2i = ___drcp_rn(s * s); 
      wght = __ldg(&CUDA_Weight[lnp2]);
      dy = __ldg(&CUDA_brightness[lnp2]) - ymod;
      
      int ixx = jp + (threadIdx.x + 1) * Lpoints1; // ZZZ, bad, strided read, BAD!
      double * __restrict__ c = &(dytemp[ixx]); //  bad c
      l = threadIdx.x + 1;
#pragma unroll 4
      while(l <= ma - CUDA_BLOCK_DIM)
	{
	  double a, b;
	  a = __ldca(c);
	  c += CUDA_BLOCK_DIM * Lpoints1;
	  b = __ldca(c);
	  c += CUDA_BLOCK_DIM * Lpoints1;
	  dyda[l-1] = a;
	  dyda[l-1 + CUDA_BLOCK_DIM] = b;
	  l += 2*CUDA_BLOCK_DIM;
	}
#pragma unroll 1
      while(l <= ma)
	{
	  dyda[l-1] = __ldca(c);
	  l += CUDA_BLOCK_DIM;
	  c += CUDA_BLOCK_DIM * Lpoints1;
	}
      
      __syncwarp();
      
      j = 0;
      double sig2iwght = sig2i * wght;
      
#pragma unroll 4
      for(l = 1; l <= lastone; l++)
	{
	  j++;
	  wt = dyda[l-1] * sig2iwght;
	  int xx = threadIdx.x + 1;
#pragma unroll 2
	  while(xx <= l)
	    {
	      alpha[j * mf1 + xx] += wt * dyda[xx-1];
	      xx += CUDA_BLOCK_DIM;
	    } /* m */
	  //__syncthreads();
	  if(threadIdx.x == 0)
	    {
	      beta[j] = beta[j] + dy * wt;
	    }
	  //__syncthreads();
	} /* l */
      
#pragma unroll 4
      for(; l <= lma; l++)
	{
	  if(iapp[l])
	    {
	      j++;
	      wt = dyda[l-1] * sig2iwght;
	      int xx = threadIdx.x + 1;
#pragma unroll 2
	      while(xx <= lastone)
		{
		  //if(xx != 0)
		  alpha[j * mf1 + xx] += wt * dyda[xx-1];
		  xx += CUDA_BLOCK_DIM;
		} /* m */
	      //__syncthreads();
	      if(threadIdx.x == 0)
		{
		  k = lastone;
		  m = lastone + 1;
		  int * __restrict__ iap = iapp + m;
		  double * __restrict__ alp = alpha + j * mf1 + k;
#pragma unroll 4
		  for(; m <= l; m++)
		    {
		      if(*iap)
			{
			  alp++;
			  __stwb(alp, *alp + wt * dyda[m-1]);
			}
		      iap++;
		    } /* m */
		  beta[j] = beta[j] + dy * wt;
		}
	      //__syncthreads();
	    }
	} /* l */
      ltrial_chisq = ltrial_chisq + dy * dy * sig2iwght;
    } /* jp */

  if(threadIdx.x == 0)
    {
      npg2[bid] = lnp2;
      trial_chisqg[bid] = ltrial_chisq;
    }
}

// SLOW (only 3 threads participate -> 1/10 perf))
  __device__ void __forceinline__ MrqcofCurve23I1IA0(freq_context * __restrict__ CUDA_LCC, double * __restrict__ alpha, double * __restrict__ beta, int bid)
{
  int lpoints = 3;
  int mf1 = CUDA_mfit1;
  int l, jp, j, k, m, lnp1, lnp2, Lpoints1 = lpoints + 1;
  double dy, sig2i, wt, ymod, coef1, coef, wght, ltrial_chisq;
  __shared__ double dydat[3][N80];
  
  lnp1 = npg1[bid] + threadIdx.x + 1;

  int ma = CUDA_ma;
  //int bid = blockIdx();
  jp = threadIdx.x + 1;
  double rave = ___drcp_rn(aveg[bid]);
  double * __restrict__ dytmpp = CUDA_LCC->dytemp, * __restrict__ cuda_sig = CUDA_sig, * __restrict__ ytemp = CUDA_LCC->ytemp;
  double * __restrict__ cuda_weight = CUDA_Weight, * __restrict__ cuda_brightness = CUDA_brightness;
  //double * __restrict__ dave = CUDA_LCC->dave;
  double * __restrict__ davep = &(dave[bid][0]);
  long int lpadd = sizeof(double) * Lpoints1;
  
  //#pragma unroll 
  if(jp <= lpoints)
    {
      int ixx = jp + Lpoints1;
      // Set the size scale coeff. deriv. explicitly zero for relative lcurves 
      dytmpp[ixx] = 0; // YYY, good, consecutive
      coef = cuda_sig[lnp1] * lpoints * rave; // / CUDA_LCC->ave;
      
      double yytmp = ytemp[jp];
      coef1 = yytmp * rave; // / CUDA_LCC->ave;
      ytemp[jp] = coef * yytmp;
      
      ixx += Lpoints1;
      double * __restrict__ dyp = dytmpp + ixx; //&(CUDA_LCC->dytemp[ixx]);
      double * __restrict__ dap = &(davep[2]);
#pragma unroll 
      for(l = 2; l <= ma - (UNRL - 1); l += UNRL, ixx += UNRL * Lpoints1)
	{
	  double dd[UNRL], dy[UNRL];
	  int ii;
	  double * __restrict__ dypp = dyp;
	  for(ii = 0; ii < UNRL; ii++)
	    {
	      dy[ii] = __ldg(dypp);
	      //dypp += Lpoints1;
	      dypp = (double *)(((char *)dypp) + lpadd);

	      dd[ii] = __ldca(dap);
	      dap++;
	    }
	  for(ii = 0; ii < UNRL; ii++)
	    {
	      __stwb(dyp, coef * (dy[ii] - coef1 * dd[ii])); //WXX
	      //dyp += Lpoints1;
	      dyp = (double *)(((char *)dyp) + lpadd);
	    }
	}
#pragma unroll 
      for(; l <= ma; l++, dyp += Lpoints1, dap++)
	__stwb(dyp, coef * ( __ldg(dyp) - coef1 * __ldca(dap))); //WXX
	//*dyp = __ldg(dyp) * coef - coef1 * __ldg(dap);

      jp += CUDA_BLOCK_DIM;
      lnp1 += CUDA_BLOCK_DIM;
    }

  __syncwarp();

  if(threadIdx.x == 0)
    {
      npg1[bid] += lpoints;
    }

  lnp2 = npg2[bid];
  ltrial_chisq = trial_chisqg[bid];
  int lastone = CUDA_lastone;
  int * __restrict__ iapp = CUDA_ia;

#pragma unroll 
  for(jp = 1; jp <= lpoints; jp++)
    {
      if(jp == 1)
	{
	  int ixx = jp + (threadIdx.x + 1) * Lpoints1; // RXX bad, strided read, BAD
	  double * __restrict__ c = dytmpp + ixx;  //&(CUDA_LCC->dytemp[ixx]);
	  l = threadIdx.x;
#pragma unroll 2
	  while(l < ma)
	    {
	      dydat[0][l] = c[0]; // YYYY RXX
	      dydat[1][l] = c[1]; // YYYY
	      dydat[2][l] = c[2]; // YYYY
	      l += CUDA_BLOCK_DIM;
	      c += CUDA_BLOCK_DIM * Lpoints1;
	    }
	  __syncwarp();
	}
      
      double * __restrict__ dyda = &dydat[jp-1][0];
      
      lnp2++;
      double s = cuda_sig[lnp2];
      ymod = ytemp[jp];
      sig2i = ___drcp_rn(s * s);
      wght = cuda_weight[lnp2];
      dy = cuda_brightness[lnp2] - ymod;
      
      j = 0;
      double sig2iwght = sig2i * wght;

      double * __restrict__ dydap = dyda + 1;
#pragma unroll 
      for(l = 2; l <= lastone; l++)
	{
	  j++;
	  wt = *dydap * sig2iwght;
	  dydap++;
	  
	  int xx = threadIdx.x + 1;
	  double * __restrict__ alp = &(alpha[j * mf1 - 1 + xx]);
#pragma unroll 2
	  while(xx <= l)
	    {
	      //if(xx != 0)
	      double * __restrict__ alp2 = alp;
	      __stwb(alp, *alp2 + wt * dyda[xx-1]);
	      xx += CUDA_BLOCK_DIM;
	      alp += CUDA_BLOCK_DIM;
	    } /* m */
	  //__syncthreads();
	  if(threadIdx.x == 0)
	    {
	      beta[j] += dy * wt;
	    }
	  //__syncthreads();
	} /* l */
      
#pragma unroll 
      for(; l <= CUDA_lastma; l++)
	{
	  if(iapp[l])
	    {
	      j++;
	      wt = *dydap * sig2iwght;
	      
	      int xx = threadIdx.x + 1;
	      double * __restrict__ alp = &alpha[j * mf1 - 1];
#pragma unroll 2
	      while(xx <= lastone)
		{
		  //if(xx != 0)
		  double const * __restrict__ alp2 = alp;
		  __stwb(alp, *alp2 + wt * dyda[xx-1]);
		  xx += CUDA_BLOCK_DIM;
		  alp += CUDA_BLOCK_DIM;
		} /* m */
	      //__syncthreads();
	      if(threadIdx.x == 0)
		{
		  k = lastone - 1;
		  m = lastone + 1;
		  int * __restrict__ iap = iapp + m;
		  double * __restrict__ alp = alpha + j * mf1 + k;
#pragma unroll 4
		  for(; m <= l; m++)
		    {
		      if(*iap)
			{
			  //k++;
			  alp++;
			  double const * __restrict__ alp2 = alp;
			  __stwb(alp, *alp2 + wt * dyda[m - 1]);
			}
		      iap++;
		    } /* m */
		  beta[j] = beta[j] + dy * wt;
		}
	      //__syncthreads();
	    }
	} /* l */
      ltrial_chisq = ltrial_chisq + dy * dy * sig2iwght;
    } /* jp */

  if(threadIdx.x == 0)
    {
      npg2[bid] = lnp2;
      trial_chisqg[bid] = ltrial_chisq;
    }
}

__device__ void __forceinline__ MrqcofCurve23I1IA1(freq_context * __restrict__ CUDA_LCC, double * __restrict__ alpha, double * __restrict__ beta, int bid)
{
  int lpoints = 3;
  int mf1 = CUDA_mfit1;
  //int bid = blockIdx();
  int l, jp, j, k, m, lnp1, lnp2, Lpoints1 = lpoints + 1;
  double dy, sig2i, wt, ymod, coef1, coef, wght, ltrial_chisq;
  __shared__ double dyda[N80];
  
  lnp1 = npg1[bid] + 1;

  int ma = CUDA_ma;
  double rave = ___drcp_rn(aveg[bid]);
  double * __restrict__ dytemp = CUDA_LCC->dytemp, * __restrict__ ytemp = CUDA_LCC->ytemp;
  
#pragma unroll 
  for(jp = 1; jp <= lpoints; jp++, lnp1++)
    {
      int ixx = jp + Lpoints1;
      // Set the size scale coeff. deriv. explicitly zero for relative lcurves 
      dytemp[ixx] = 0; // YYY, good?, same for all threads??
      coef = __ldg(&CUDA_sig[lnp1]) * lpoints * rave; // / CUDA_LCC->ave;
      
      double yytmp = ytemp[jp];
      coef1 = yytmp * rave; // / CUDA_LCC->ave;
      ytemp[jp] = coef * yytmp;
      
      ixx += Lpoints1;
      double * __restrict__ dyp = &(dytemp[ixx]);
      double * __restrict__ dap = &(dave[bid][2]);
      l = 2 + threadIdx.x;
#pragma unroll 2
      while(l <= ma)
	{
	  double dy = __ldg(dyp);
	  double dd = __ldca(dap);
	  dap += CUDA_BLOCK_DIM;
	  __stwb(dyp, coef * (dy - coef1 * dd));
	  dyp += Lpoints1 * CUDA_BLOCK_DIM;
	  l += CUDA_BLOCK_DIM;
	  ixx += CUDA_BLOCK_DIM * Lpoints1;
	}
    }

  __syncwarp();

  if(threadIdx.x == 0)
    {
      npg1[bid] += lpoints;
    }

  lnp2 = npg2[bid];
  ltrial_chisq = trial_chisqg[bid];

  int lastone = CUDA_lastone, lma = CUDA_lastma;
  int * __restrict__ iapp = CUDA_ia;
  
#pragma unroll 
  for(jp = 1; jp <= lpoints; jp++) 
    {
      lnp2++;
      double s = __ldg(&CUDA_sig[lnp2]);
      ymod = __ldca(&(ytemp[jp]));
      sig2i = ___drcp_rn(s * s); 
      wght = __ldg(&CUDA_Weight[lnp2]);
      dy = __ldg(&CUDA_brightness[lnp2]) - ymod;
      
      int ixx = jp + (threadIdx.x + 1) * Lpoints1; // ZZZ, bad, strided read, BAD!
      double * __restrict__ c = &(dytemp[ixx]); //  bad c
      l = threadIdx.x + 1;
#pragma unroll 4
      while(l <= ma - CUDA_BLOCK_DIM)
	{
	  double a, b;
	  a = __ldca(c);
	  c += CUDA_BLOCK_DIM * Lpoints1;
	  b = __ldca(c);
	  c += CUDA_BLOCK_DIM * Lpoints1;
	  dyda[l-1] = a;
	  dyda[l-1 + CUDA_BLOCK_DIM] = b;
	  l += 2*CUDA_BLOCK_DIM;
	}
#pragma unroll 1
      while(l <= ma)
	{
	  dyda[l-1] = __ldca(c);
	  l += CUDA_BLOCK_DIM;
	  c += CUDA_BLOCK_DIM * Lpoints1;
	}
      
      __syncwarp();
      
      j = 0;
      double sig2iwght = sig2i * wght;
      
#pragma unroll 4
      for(l = 1; l <= lastone; l++)
	{
	  j++;
	  wt = dyda[l-1] * sig2iwght;
	  int xx = threadIdx.x + 1;
	  double * __restrict__ alp = &alpha[j * mf1 + xx]; 
#pragma unroll 2
	  while(xx <= l)
	    {
	      double const * __restrict alp2 = alp;
	      __stwb(alp, *alp2 +  wt * dyda[xx-1]);
	      xx += CUDA_BLOCK_DIM;
	      alp += CUDA_BLOCK_DIM;
	    } /* m */
	  //__syncthreads();
	  if(threadIdx.x == 0)
	    {
	      beta[j] = beta[j] + dy * wt;
	    }
	  //__syncthreads();
	} /* l */
      
#pragma unroll 4
      while(l <= lma)
	{
	  if(iapp[l])
	    {
	      j++;
	      wt = dyda[l-1] * sig2iwght;
	      int xx = threadIdx.x + 1;
	      double * __restrict__ alp = &alpha[j * mf1 + xx]; 
#pragma unroll 2
	      while(xx <= lastone)
		{
		  //if(xx != 0)
		  double const * __restrict alp2 = alp;
		  __stwb(alp, *alp2 + wt * dyda[xx-1]);
		  xx += CUDA_BLOCK_DIM;
		  alp += CUDA_BLOCK_DIM;
		} /* m */
	      //__syncthreads();
	      if(threadIdx.x == 0)
		{
		  k = lastone;
		  m = lastone + 1;
		  int * __restrict__ iap = iapp + m;
		  double * __restrict__ alp = alpha + j * mf1 + k;
#pragma unroll 4
		  while(m <= l)
		    {
		      if(*iap)
			{
			  alp++;
			  __stwb(alp, *alp + wt * dyda[m-1]);
			}
		      iap++;
		      m++;
		    } /* m */
		  beta[j] = beta[j] + dy * wt;
		}
	      //__syncthreads();
	    }
	  l++;
	} /* l */
      ltrial_chisq = ltrial_chisq + dy * dy * sig2iwght;
    } /* jp */
  
  if(threadIdx.x == 0)
    {
      npg2[bid] = lnp2;
      trial_chisqg[bid] = ltrial_chisq;
    }
}

__device__ void __forceinline__ MrqcofCurve2I0IA0(freq_context * __restrict__ CUDA_LCC, double * __restrict__ alpha, double * __restrict__ beta, int lpoints, int bid)
{
  //inrel = 0;
  int l, jp, j, /*k, m,*/ lnp2, Lpoints1 = lpoints + 1;
  double dy, sig2i, wt, ymod, wght, ltrial_chisq;
  int mf1 = CUDA_mfit1;
  
  __shared__ double dydat[4][N80];
  
  __syncwarp(); // remove sync ?

  if(threadIdx.x == 0)
    {
      npg1[bid] += lpoints;
    }

  lnp2 = npg2[bid];
  ltrial_chisq = trial_chisqg[bid];

  int ma = CUDA_ma, lma = CUDA_lastma;
  int lastone = CUDA_lastone;
  int * __restrict__ iapp = CUDA_ia;
  double * __restrict__ dytemp = CUDA_LCC->dytemp, * __restrict__ ytemp = CUDA_LCC->ytemp;
  
#pragma unroll 2
  for(jp = 1; jp <= lpoints; jp++)
    {
      if(((jp-1)&3) == 0)
	{
	  int tid = threadIdx.x >> 2;
	  int u = threadIdx.x & 3;
	  int ixx = jp + (tid + 1) * Lpoints1; // ZZZ bad, strided read dytemp, BAD
	  double * __restrict__ c = &(dytemp[ixx]);//, *dddc = ddd + ixx;
	  c += u;
	  l = tid;
#pragma unroll 4
	  while(l < ma)
	    {
#ifdef DYTEMP_NEW
	      dydat[u][l] = dytemp2[bid][jp][(l << 2) + u + 1];
#else
	      dydat[u][l] = __ldca(c); //*dddc //__ldca(c); // YYYY
#endif
	      l += CUDA_BLOCK_DIM/4;
	      c += CUDA_BLOCK_DIM/4 * Lpoints1;
	    }
	}
      __syncwarp();
      
      double * __restrict__ dyda = &(dydat[(jp-1) & 3][0]);	  

      /*
      int ixx = jp + (threadIdx.x + 1) * Lpoints1; // ZZZ bad, strided read, BAD
      double *c = &(CUDA_LCC->dytemp[ixx]);
#pragma unroll 2
      for(l = threadIdx.x; l < ma; l += CUDA_BLOCK_DIM, c += CUDA_BLOCK_DIM * Lpoints1)
	dyda[l] = __ldca(c); // YYYY
      */  

      lnp2++;
      double s = __ldg(&CUDA_sig[lnp2]);
      ymod = __ldca(&ytemp[jp]);
      sig2i = ___drcp_rn(s * s);
      wght = __ldg(&CUDA_Weight[lnp2]);
      dy = __ldg(&CUDA_brightness[lnp2]) - ymod;

      j = 0;
      double sig2iwght = sig2i * wght;

#pragma unroll 2
      for(l = 2; l <= lastone; l++)
	{
	  j++;
	  wt = dyda[l-1] * sig2iwght;

	  int xx = threadIdx.x + 1;
	  double * __restrict__ alph = (&alpha[j * mf1 - 1]) + xx;
#pragma unroll 2
	  while(xx <= l)
	    {
	      //if(xx != 0)
	      //alpha[j * mf1 - 1 + xx] += wt * dyda[xx-1];
	      double const * __restrict__ alph2 = alph;
	      __stwb(alph, __ldca(alph2) + wt * dyda[xx-1]); //ldg
	      //*alpha += wt * dyda[xx-1];
	      //alpha += CUDA_BLOCK_DIM;
	      //xx += CUDA_BLOCK_DIM;
	      alph  += CUDA_BLOCK_DIM;
	    } /* m */
	  //__syncthreads();
	  if(threadIdx.x == 0)
	    {
	      beta[j] = beta[j] + dy * wt;
	    }
	  //__syncthreads();
	  l++;
	} /* l */
	  
#pragma unroll 1
      while(l <= lma)
	{
	  if(iapp[l])
	    {
	      j++;
	      wt = dyda[l-1] * sig2iwght;

	      int xx = threadIdx.x + 1;
	      double * __restrict__ alph = &alpha[j * mf1 - 1 + xx]; // + xx;
#pragma unroll 2
	      while(xx <= lastone)
		{
		  //if(xx != 0)
		  double const * __restrict__ alph2 = alph;
		  __stwb(alph, __ldca(alph2) + wt * dyda[xx-1]); //ldg
		  //*alpha += wt * dyda[xx-1];
		  //alpha += CUDA_BLOCK_DIM;
		  alph  += CUDA_BLOCK_DIM;
		} /* m */
	      //__syncthreads();
	      if(threadIdx.x == 0)
		{
		  int k = lastone - 1;
		  int m = lastone + 1;
		  int * __restrict__ iap = iapp + m;
		  double * __restrict__ alp = alpha + j * mf1 + k;
		  beta[j] = beta[j] + dy * wt;
#pragma unroll 4
		  while(m <= l)
		    {
		      if(*iap)
			{
			  //k++;
			  alp++;
			  double const * __restrict__ alp2 = alp;
			  __stwb(alp, __ldca(alp2) + wt * dyda[m - 1]);
			}
		      iap++;
		      m++;
		    } /* m */
		}
	      //__syncthreads();
	    }
	  l++;
	} /* l */
      ltrial_chisq = ltrial_chisq + dy * dy * sig2iwght;
    } /* jp */

  if(threadIdx.x == 0)
    {
      npg2[bid] = lnp2;
      trial_chisqg[bid] = ltrial_chisq;
    }
}

// SLOWW
__device__ void __forceinline__ MrqcofCurve2I1IA0(freq_context *__restrict__ CUDA_LCC, double * __restrict__ alpha, double * __restrict__ beta, int lpoints, int bid)
{
  int l, jp, j, k, m, lnp1, lnp2, Lpoints1 = lpoints + 1;
  double dy, sig2i, wt, ymod, coef1, coef, wght, ltrial_chisq;
  int mf1 = CUDA_mfit1;
  __shared__ double dydat[4][N80];
  //__shared__ double ddd[4500];
  
  lnp1 = npg1[bid] + threadIdx.x + 1;

  int ma = CUDA_ma;
  //int bid = blockIdx();
  jp = threadIdx.x + 1;
  double rave = ___drcp_rn(aveg[bid]);
  double * __restrict__ dytempp = CUDA_LCC->dytemp, * __restrict__ ytempp = CUDA_LCC->ytemp;
  double * __restrict__ cuda_sig = CUDA_sig;
  //double * __restrict__ davep = CUDA_LCC->dave;
  double * __restrict__ davep = &(dave[bid][0]);
  long int lpadd = sizeof(double) * Lpoints1;
  
#pragma unroll 1
  while(jp <= lpoints)
    {
      int ixx = jp + Lpoints1;
      // Set the size scale coeff. deriv. explicitly zero for relative lcurves 
      dytempp[ixx] = 0; // YYY, good, consecutive

      //ddd[ixx] = 0;
      coef = __ldca(&cuda_sig[lnp1]) * lpoints * rave; // / CUDA_LCC->ave;
      
      double yytmp = __ldca(&ytempp[jp]);
      coef1 = yytmp * rave; // / CUDA_LCC->ave;
      ytempp[jp] = coef * yytmp;
      
      ixx += Lpoints1;
      double * __restrict__ dyp = &(dytempp[ixx]), *__restrict__ dypp; //, *ddyp = ddd + ixx, *ddypp; 
      double * __restrict__ dap = &(davep[2]);
      //dypp = dyp;

#pragma unroll 1
      for(l = 2; l <= ma - (2 - 1); l += 2, ixx += 2 * Lpoints1)
	{
	  double dd[2], dy[2];
	  int ii;
	  dypp = dyp;
	  //ddypp = ddyp;
	  for(ii = 0; ii < 2; ii++)
	    {
	      dy[ii] = __ldca(dypp);
	      //dypp += Lpoints1;
	      dypp = (double *)(((char *)dypp) + lpadd);

	      //ddyp += Lpoints1;
	      dd[ii] = __ldca(dap);
	      dap++;
	    }
	  for(ii = 0; ii < 2; ii++)
	    {
	      double d = coef * (dy[ii] - coef1 * dd[ii]);
	      *dyp = d;
	      //dyp += Lpoints1;
	      dyp = (double *)(((char *)dyp) + lpadd);

	      //*ddypp = d;
	      //ddypp += Lpoints1;
	    }
	}
#pragma unroll 
      while(l <= ma)
	{
	  double d = coef * __ldca(&dyp[0]) - coef1 * __ldca(&dap[0]);
	  *dyp = d;
	  l++;
	  //dyp += Lpoints1;
	  dyp = (double *)(((char *)dyp) + lpadd);

	  dap++;
	}
      jp += CUDA_BLOCK_DIM;
      lnp1 += CUDA_BLOCK_DIM;
    }
  
  __syncwarp();
  
  if(threadIdx.x == 0)
    {
      npg1[bid] += lpoints;
    }
  
  lnp2 = npg2[bid];
  ltrial_chisq = trial_chisqg[bid];
  
  int lastone = CUDA_lastone, lma = CUDA_lastma;
  int * __restrict__ iapp = CUDA_ia;
  double * __restrict__ cuda_weight = CUDA_Weight, * __restrict__ cuda_brightness = CUDA_brightness;
  
#pragma unroll 4
  for(jp = 1; jp <= lpoints; jp++)
    {
      if(((jp-1)&3) == 0)
	{
	  int tid = threadIdx.x >> 2;
	  int u = threadIdx.x & 3;
	  int ixx = jp + (tid + 1) * Lpoints1; // ZZZ bad, strided read dytemp, BAD
	  double * __restrict__ c = &(dytempp[ixx]);//, *dddc = ddd + ixx;
	  c += u;
	  l = tid;
#pragma unroll 4
	  while(l < ma)
	    {
#ifdef DYTEMP_NEW
	      dydat[u][l] = dytemp2[bid][jp][(l << 2) + u + 1];
#else
	      dydat[u][l] = __ldca(c); //*dddc //__ldca(c); // YYYY
#endif
	      l += CUDA_BLOCK_DIM/4;
	      c += CUDA_BLOCK_DIM/4 * Lpoints1;
	    }
	}
      __syncwarp();
      double * __restrict__ dyda = &(dydat[(jp-1) & 3][0]);	  
      lnp2++;
      double s = cuda_sig[lnp2];
      ymod = ytempp[jp];
      sig2i = ___drcp_rn(s * s);
      wght = cuda_weight[lnp2];
      dy = cuda_brightness[lnp2] - ymod;

      j = 0;
      double sig2iwght = sig2i * wght;


#pragma unroll 2
      for(l = 2; l <= lastone; l++)
	{
	  j++;
	  wt = dyda[l-1] * sig2iwght;

	  int xx = threadIdx.x;
	  double *__restrict__ alph = &alpha[j * mf1 + xx];
#pragma unroll 2
	  while(xx < l)
	    {
	      //if(xx != 0)
	      __stwb(alph, __ldca(alph) +  wt * dyda[xx]);
	      alph += CUDA_BLOCK_DIM;
	      xx += CUDA_BLOCK_DIM;
	    } /* m */
	  //__syncthreads();
	  if(threadIdx.x == 0)
	    {
	      beta[j] = beta[j] + dy * wt;
	    }
	  //	  __syncthreads();
	} /* l */
	  
#pragma unroll 1
      for(; l <= lma; l++)
	{
	  if(iapp[l])
	    {
	      j++;
	      wt = dyda[l-1] * sig2iwght;

	      int xx = threadIdx.x;
	      double * __restrict__ alph = &alpha[j * mf1 + xx];
#pragma unroll 2
	      while(xx < lastone)
		{
		  //if(xx != 0)
		  __stwb(alph, __ldca(alph) + wt * dyda[xx]);
		  alph += CUDA_BLOCK_DIM;
		  xx += CUDA_BLOCK_DIM;
		} /* m */
	      //__syncthreads();
	      if(threadIdx.x == 0)
		{
		  k = lastone - 1;
		  m = lastone + 1;
		  int * __restrict__ iap = iapp + m;
		  double * __restrict__ alp = &(alpha[j * mf1 + k]);
#pragma unroll 4
		  for(; m <= l; m++)
		    {
		      if(*iap)
			{
			  //k++;
			  ++alp;
			  __stwb(alp, __ldca(alp) + wt * dyda[m-1]);
			}
		      iap++;
		    } /* m */
		  beta[j] +=  dy * wt;
		}
	      //__syncthreads();
	    }
	} /* l */
      ltrial_chisq = ltrial_chisq + dy * dy * sig2iwght;
    } /* jp */

  if(threadIdx.x == 0)
    {
      npg2[bid] = lnp2;
      trial_chisqg[bid] = ltrial_chisq;
    }
}

__device__ void __forceinline__ MrqcofCurve2I0IA1(freq_context * __restrict__ CUDA_LCC, double * __restrict__ alpha, double * __restrict__ beta, int lpoints, int bid)
{
  int l, jp, j, k, m, lnp2, Lpoints1 = lpoints + 1;
  double dy, sig2i, wt, ymod, wght, ltrial_chisq;
  int mf1 = CUDA_mfit1;
  __shared__ double dyda[N80];
  
  __syncwarp(); // remove

  if(threadIdx.x == 0)
    {
      npg1[bid] += lpoints;
    }

  lnp2 = npg2[bid];
  ltrial_chisq = trial_chisqg[bid];

  int ma = CUDA_ma, lma = CUDA_lastma;
  int lastone = CUDA_lastone;
  int * __restrict__ iapp = CUDA_ia;
  double * __restrict__ dytemp = CUDA_LCC->dytemp, *ytemp = CUDA_LCC->ytemp;
  
#pragma unroll 2
  for(jp = 1; jp <= lpoints; jp++) // CHANGE LOOP threadIdx.x ?
    {
      lnp2++;
      double s = __ldg(&CUDA_sig[lnp2]);
      ymod = __ldca(&(ytemp[jp]));
      sig2i = ___drcp_rn(s * s); 
      wght = __ldg(&CUDA_Weight[lnp2]);
      dy = __ldg(&CUDA_brightness[lnp2]) - ymod;

      int ixx = jp + (threadIdx.x + 1) * Lpoints1; // ZZZ, bad, strided read, BAD!
      double * __restrict__ c = &(dytemp[ixx]); //  bad c
      l = threadIdx.x + 1;
#pragma unroll 2
      while(l <= ma - CUDA_BLOCK_DIM)
	{
	  double a, b;
#ifdef DYTEMP_NEW
	  a = dytemp2[bid][jp][l];
#else
	  a = __ldca(c);
#endif
	  c += CUDA_BLOCK_DIM * Lpoints1;
#ifdef DYTEMP_NEW
	  b = dytemp2[bid][jp][l + CUDA_BLOCK_DIM];
#else
	  b = __ldca(c);
#endif
	  c += CUDA_BLOCK_DIM * Lpoints1;
	  dyda[l-1] = a;
	  dyda[l-1 + CUDA_BLOCK_DIM] = b;
	  l += 2*CUDA_BLOCK_DIM;
	}
      //#pragma unroll 1
      //for( ; l <= ma; l += CUDA_BLOCK_DIM, c += CUDA_BLOCK_DIM * Lpoints1)
      if(l <= ma)
#ifdef DYTEMP_NEW
	dyda[l - 1] = dytemp2[bid[jp][l];
#else
	dyda[l-1] = __ldca(c);
#endif
	    
      __syncwarp();

      j = 0;
      double sig2iwght = sig2i * wght;

#pragma unroll 2
      for(l = 1; l <= lastone; l++)
	{
	  j++;
	  wt = dyda[l-1] * sig2iwght;
	  int xx = threadIdx.x + 1;
	  double * __restrict__ alp = alpha + mf1 + xx;
#pragma unroll 2
	  while(xx <= l)
	    {
	      double const * __restrict__ alp2 = alp;
	      __stwb(alp, __ldca(alp2) + wt * dyda[xx-1]);
	      alp += mf1;
	      xx += CUDA_BLOCK_DIM;
	    } /* m */
	  //__syncthreads();
	  if(threadIdx.x == 0)
	    {
	      beta[j] = beta[j] + dy * wt;
	    }
	  //__syncthreads();
	} /* l */
	  
#pragma unroll 2
      for(; l <= lma; l++)
	{
	  if(iapp[l])
	    {
	      j++;
	      wt = dyda[l-1] * sig2iwght;
	      int xx = threadIdx.x + 1;
	      double * __restrict__ alp = &alpha[j * mf1 + xx];
#pragma unroll 2
	      while(xx <= lastone)
		{
		  //if(xx != 0)
		  //alpha[j * mf1 + xx] += wt * dyda[xx-1];
		  double const * __restrict__ alp2 = alp;
		  __stwb(alp, __ldca(alp2) + wt * dyda[xx-1]);
		  xx += CUDA_BLOCK_DIM;
		  alp  += CUDA_BLOCK_DIM;
		} /* m */
	      //__syncthreads();
	      if(threadIdx.x == 0)
		{
		  k = lastone;
		  m = lastone + 1;
		  int * __restrict__ iap = iapp + m;
		  double * __restrict__ alp = alpha + j * mf1 + k;
#pragma unroll 4
		  for(; m <= l; m++)
		    {
		      if(*iap)
			{
			  alp++;
			  __stwb(alp, __ldca(alp) + wt * dyda[m-1]);
			}
		      iap++;
		    } /* m */
		  beta[j] = beta[j] + dy * wt;
		}
	      //__syncthreads();
	    }
	} /* l */
      ltrial_chisq = ltrial_chisq + dy * dy * sig2iwght;
    } /* jp */

  if(threadIdx.x == 0)
    {
      npg2[bid] = lnp2;
      trial_chisqg[bid] = ltrial_chisq;
    }
}

// WORKING, SLOW
  __device__ void __forceinline__ MrqcofCurve2I1IA1(freq_context * __restrict__ CUDA_LCC, double * __restrict__ alpha, double * __restrict__ beta, int lpoints, int bid)
{
  int l, jp, j, k, m, lnp1, lnp2, Lpoints1 = lpoints + 1;
  double dy, sig2i, wt, ymod, coef1, coef, wght, ltrial_chisq;
  int mf1 = CUDA_mfit1;
  __shared__ double dyda[N80];
  
  lnp1 = npg1[bid] + threadIdx.x + 1;
  
  int ma = CUDA_ma;
  //int bid = blockIdx();
  jp = threadIdx.x + 1;
  double rave = ___drcp_rn(aveg[bid]);
  double * __restrict__ dytemp = CUDA_LCC->dytemp, * __restrict__ ytemp = CUDA_LCC->ytemp;
  
#pragma unroll 1
  while(jp <= lpoints)
    {
      int ixx = jp + Lpoints1;
      // Set the size scale coeff. deriv. explicitly zero for relative lcurves 
      dytemp[ixx] = 0; // YYY, good, consecutive
      coef = __ldg(&CUDA_sig[lnp1]) * lpoints * rave; // / CUDA_LCC->ave;

      double yytmp = ytemp[jp];
      coef1 = yytmp * rave; // / CUDA_LCC->ave;
      ytemp[jp] = coef * yytmp;

      ixx += Lpoints1;
      double * __restrict__ dyp = &(dytemp[ixx]);
      double * __restrict__ dap = &(dave[bid][2]);
#pragma unroll 2
      for(l = 2; l <= ma - (UNRL - 1); l += UNRL, ixx += UNRL * Lpoints1)
	{
	  double dd[UNRL], dy[UNRL];
	  int ii;
	  double * __restrict__ dypp = dyp;
#pragma unroll 4
	  for(ii = 0; ii < UNRL; ii++)
	    {
	      dy[ii] = __ldca(dyp);
	      dyp += Lpoints1;
	      dd[ii] = __ldg(dap);
	      dap++;
	    }
#pragma unroll 4
	  for(ii = 0; ii < UNRL; ii++)
	    {
	      __stwb(dypp, coef * (dy[ii] - coef1 * dd[ii]));
	      dypp += Lpoints1;
	    }
	}
#pragma unroll 3
      for(; l <= ma; l++, dyp += Lpoints1, dap++)
	__stwb(dyp, __ldca(dyp) * coef - coef1 * __ldg(dap));
      
      jp += CUDA_BLOCK_DIM;
      lnp1 += CUDA_BLOCK_DIM;
    }

  __syncwarp();

  if(threadIdx.x == 0)
    {
      npg1[bid] += lpoints;
    }

  lnp2 = npg2[bid];
  ltrial_chisq = trial_chisqg[bid];

  int lastone = CUDA_lastone, lma = CUDA_lastma;
  int * __restrict__ iapp = CUDA_ia;
  
#pragma unroll 2
  for(jp = 1; jp <= lpoints; jp++) // CHANGE LOOP threadIDx.x ?
    {
      lnp2++;
      double s = __ldg(&CUDA_sig[lnp2]);
      ymod = __ldca(&(ytemp[jp]));
      sig2i = ___drcp_rn(s * s); 
      wght = __ldg(&CUDA_Weight[lnp2]);
      dy = __ldg(&CUDA_brightness[lnp2]) - ymod;

      int ixx = jp + (threadIdx.x + 1) * Lpoints1; // ZZZ, bad, strided read, BAD!
      double * __restrict__ c = &(dytemp[ixx]); //  bad c
      l = threadIdx.x + 1;
#pragma unroll 2
      while(l <= ma - CUDA_BLOCK_DIM)
	{
	  double a, b;
#ifdef DYTEMP_NEW
	  a = dytemp2[bid][jp][l];
#else
	  a = __ldca(c);
#endif
	  c += CUDA_BLOCK_DIM * Lpoints1;
#ifdef DYTEMP_NEW
	  b = dytemp2[bid][jp][l + CUDA_BLOCK_DIM];
#else
	  b = __ldca(c);
#endif
	  c += CUDA_BLOCK_DIM * Lpoints1;
	  dyda[l-1] = a;
	  dyda[l-1 + CUDA_BLOCK_DIM] = b;
	  l += 2*CUDA_BLOCK_DIM;
	}
      //#pragma unroll 2
      //for( ; l <= ma; l += CUDA_BLOCK_DIM, c += CUDA_BLOCK_DIM * Lpoints1)
      if(l < ma)
#ifdef DYTEMP_NEW
	dyda[l - 1] = dytemp2[bid][jp][l];
#else
	dyda[l-1] = __ldca(c);
#endif
	    
      __syncwarp();

      j = 0;
      double sig2iwght = sig2i * wght;

#pragma unroll 2
      for(l = 1; l <= lastone; l++)
	{
	  j++;
	  wt = dyda[l-1] * sig2iwght;
	  int xx = threadIdx.x + 1;
	  double * __restrict__ alp = &alpha[j * mf1 + xx];
#pragma unroll 2
	  while(xx <= l)
	    {
	      double const * __restrict__ alp2 = alp;
	      __stwb(alp, *alp2 + wt * dyda[xx-1]);
	      xx += CUDA_BLOCK_DIM;
	      alp += CUDA_BLOCK_DIM;
	    } // m 
	  //__syncthreads();
	  if(threadIdx.x == 0)
	    {
	      beta[j] = beta[j] + dy * wt;
	    }
	  //__syncthreads();
	} // l
	  
#pragma unroll 1
      for(; l <= lma; l++)
	{
	  if(iapp[l])
	    {
	      j++;
	      wt = dyda[l-1] * sig2iwght;
	      int xx = threadIdx.x + 1;
	      double * __restrict__ alp = &alpha[j * mf1 + xx];
#pragma unroll 2
	      while(xx <= lastone)
		{
		  //if(xx != 0)
		  double const * __restrict__ alp2 = alp;
		  //alpha[j * mf1 + xx] += wt * dyda[xx-1];
		  *alp = *alp2 + wt * dyda[xx-1];
		  xx += CUDA_BLOCK_DIM;
		} // m 
	      //__syncthreads();
	      if(threadIdx.x == 0)
		{
		  k = lastone;
		  m = lastone + 1;
		  int * __restrict__ iap = iapp + m;
		  double * __restrict__ alp = alpha + j * mf1 + k;
#pragma unroll 4
		  for(; m <= l; m++)
		    {
		      if(*iap)
			{
			  alp++;
			  double const * __restrict__ alp2 = alp;
			  *alp = *alp2 + wt * dyda[m-1];
			}
		      iap++;
		    } // m 
		  beta[j] = beta[j] + dy * wt;
		}
	      //__syncthreads();
	    }
	} // l 
      ltrial_chisq = ltrial_chisq + dy * dy * sig2iwght;
    } // jp 

  if(threadIdx.x == 0)
    {
      npg2[bid] = lnp2;
      trial_chisqg[bid] = ltrial_chisq;
    }
}

__global__ void
__launch_bounds__(768) 
CudaCalculateIter1Mrqcof1Curve2I0IA0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I0IA0(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}


__global__ void
__launch_bounds__(768) 
CudaCalculateIter1Mrqcof1Curve2I0IA1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I0IA1(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}


__global__ void
__launch_bounds__(768) 
CudaCalculateIter1Mrqcof1Curve2I1IA0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I1IA0(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}

__global__ void
__launch_bounds__(768) 
CudaCalculateIter1Mrqcof1Curve2I1IA1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I1IA1(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}

__global__ 
void CudaCalculateIter1Mrqcof2CurveM12I0IA1(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  double *atryp = atry[bid]; //CUDA_LCC->atry;
  mrqcof_matrix(CUDA_LCC, atryp, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, atryp, 0, lpoints, bid);
  MrqcofCurve2I0IA1(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, lpoints, bid);
}

__global__ 
void CudaCalculateIter1Mrqcof2CurveM12I0IA0(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  double *atryp = atry[bid]; //CUDA_LCC->atry;
  mrqcof_matrix(CUDA_LCC, atryp, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, atryp, 0, lpoints, bid);
  MrqcofCurve2I0IA0(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, lpoints, bid);
}

__global__ void CudaCalculateIter1Mrqcof2Curve2I0IA0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  MrqcofCurve23I0IA0(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, bid);
}

__global__ void CudaCalculateIter1Mrqcof2Curve2I0IA1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  MrqcofCurve23I0IA1(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, bid);
}


// SLOW
__global__ void CudaCalculateIter1Mrqcof2Curve2I1IA0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  MrqcofCurve23I1IA0(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, bid);
}



__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof2Curve2I1IA1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I1IA1(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, bid);
}



__global__ 
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1CurveM12I0IA0(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  double *cg = CUDA_LCC->cg;
  mrqcof_matrix(CUDA_LCC, cg, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, cg, 0, lpoints, bid);
  MrqcofCurve2I0IA0(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, lpoints, bid);
}


__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1CurveM12I0IA1(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  double *cg = CUDA_LCC->cg;
  mrqcof_matrix(CUDA_LCC, cg, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, cg, 0, lpoints, bid);
  MrqcofCurve2I0IA1(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, lpoints, bid);
}



__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1CurveM12I1IA0(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  double *cg = CUDA_LCC->cg;
  mrqcof_matrix(CUDA_LCC, cg, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, cg, 1, lpoints, bid);
  MrqcofCurve2I1IA0(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, lpoints, bid);
}


__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1CurveM12I1IA1(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  double *cg = CUDA_LCC->cg;
  mrqcof_matrix(CUDA_LCC, cg, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, cg, 1, lpoints, bid);
  MrqcofCurve2I1IA1(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, lpoints, bid);
}


__global__ 
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1Curve1LastI0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  if(CUDA_LCC->ytemp == NULL) return;

  mrqcof_curve1_lastI0(CUDA_LCC, CUDA_LCC->cg, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}


__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1Curve1LastI1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  mrqcof_curve1_lastI1(CUDA_LCC, CUDA_LCC->cg, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}

__global__ 
void CudaCalculateIter1Mrqcof2CurveM12I1IA1(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  double *atryp = atry[bid]; //CUDA_LCC->atry;
  mrqcof_matrix(CUDA_LCC, atryp, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, atryp, 1, lpoints, bid);
  MrqcofCurve2I1IA1(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, lpoints, bid);
}

__global__ 
__launch_bounds__(384) 
void CudaCalculateIter1Mrqcof2CurveM12I1IA0(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  double *atryp = atry[bid]; //CUDA_LCC->atry;
  mrqcof_matrix(CUDA_LCC, atryp, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, atryp, 1, lpoints, bid);
  MrqcofCurve2I1IA0(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, lpoints, bid);
}

__global__
__launch_bounds__(768) 
void CudaCalculateIter1Mrqcof2Curve1LastI0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  mrqcof_curve1_lastI0(CUDA_LCC, atry[bid], CUDA_LCC->covar, CUDA_LCC->da, bid);
}

__global__
__launch_bounds__(1024) 
void CudaCalculateIter1Mrqcof2Curve1LastI1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  mrqcof_curve1_lastI1(CUDA_LCC, atry[bid], CUDA_LCC->covar, CUDA_LCC->da, bid);
}