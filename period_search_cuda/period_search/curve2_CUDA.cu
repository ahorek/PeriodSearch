#include "hip/hip_runtime.h"
//#ifndef __HIPCC__
//#define __HIPCC__
//#endif

#include <stdio.h>
#include <stdlib.h>
#include "globals_CUDA.h"
#include "declarations_CUDA.h"
//#include <hip/hip_runtime.h>
#include <>

__global__ void
__launch_bounds__(768) 
CudaCalculateIter1Mrqcof1Curve2I0IA0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I0IA0(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}


__global__ void
__launch_bounds__(768) 
CudaCalculateIter1Mrqcof1Curve2I0IA1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I0IA1(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}


__global__ void
__launch_bounds__(768) 
CudaCalculateIter1Mrqcof1Curve2I1IA0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I1IA0(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}

__global__ void
__launch_bounds__(768) 
CudaCalculateIter1Mrqcof1Curve2I1IA1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I1IA1(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}

__global__ 
void CudaCalculateIter1Mrqcof2CurveM12I0IA1(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  double *atryp = atry[bid]; //CUDA_LCC->atry;
  mrqcof_matrix(CUDA_LCC, atryp, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, atryp, 0, lpoints, bid);
  MrqcofCurve2I0IA1(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, lpoints, bid);
}

__global__ 
void CudaCalculateIter1Mrqcof2CurveM12I0IA0(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  double *atryp = atry[bid]; //CUDA_LCC->atry;
  mrqcof_matrix(CUDA_LCC, atryp, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, atryp, 0, lpoints, bid);
  MrqcofCurve2I0IA0(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, lpoints, bid);
}

__global__ void CudaCalculateIter1Mrqcof2Curve2I0IA0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  MrqcofCurve23I0IA0(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, bid);
}

__global__ void CudaCalculateIter1Mrqcof2Curve2I0IA1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  MrqcofCurve23I0IA1(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, bid);
}


// SLOW
__global__ void CudaCalculateIter1Mrqcof2Curve2I1IA0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  MrqcofCurve23I1IA0(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, bid);
}



__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof2Curve2I1IA1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I1IA1(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, bid);
}



__global__ 
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1CurveM12I0IA0(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  double *cg = CUDA_LCC->cg;
  mrqcof_matrix(CUDA_LCC, cg, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, cg, 0, lpoints, bid);
  MrqcofCurve2I0IA0(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, lpoints, bid);
}


__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1CurveM12I0IA1(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  double *cg = CUDA_LCC->cg;
  mrqcof_matrix(CUDA_LCC, cg, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, cg, 0, lpoints, bid);
  MrqcofCurve2I0IA1(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, lpoints, bid);
}



__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1CurveM12I1IA0(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  double *cg = CUDA_LCC->cg;
  mrqcof_matrix(CUDA_LCC, cg, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, cg, 1, lpoints, bid);
  MrqcofCurve2I1IA0(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, lpoints, bid);
}


__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1CurveM12I1IA1(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  double *cg = CUDA_LCC->cg;
  mrqcof_matrix(CUDA_LCC, cg, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, cg, 1, lpoints, bid);
  MrqcofCurve2I1IA1(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, lpoints, bid);
}


__global__ 
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1Curve1LastI0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  if(CUDA_LCC->ytemp == NULL) return;

  mrqcof_curve1_lastI0(CUDA_LCC, CUDA_LCC->cg, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}


__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1Curve1LastI1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  mrqcof_curve1_lastI1(CUDA_LCC, CUDA_LCC->cg, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}

__global__ 
void CudaCalculateIter1Mrqcof2CurveM12I1IA1(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  double *atryp = atry[bid]; //CUDA_LCC->atry;
  mrqcof_matrix(CUDA_LCC, atryp, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, atryp, 1, lpoints, bid);
  MrqcofCurve2I1IA1(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, lpoints, bid);
}

__global__ 
__launch_bounds__(384) 
void CudaCalculateIter1Mrqcof2CurveM12I1IA0(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  double *atryp = atry[bid]; //CUDA_LCC->atry;
  mrqcof_matrix(CUDA_LCC, atryp, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, atryp, 1, lpoints, bid);
  MrqcofCurve2I1IA0(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, lpoints, bid);
}

__global__
__launch_bounds__(768) 
void CudaCalculateIter1Mrqcof2Curve1LastI0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  mrqcof_curve1_lastI0(CUDA_LCC, atry[bid], CUDA_LCC->covar, CUDA_LCC->da, bid);
}

__global__
__launch_bounds__(1024) 
void CudaCalculateIter1Mrqcof2Curve1LastI1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  mrqcof_curve1_lastI1(CUDA_LCC, atry[bid], CUDA_LCC->covar, CUDA_LCC->da, bid);
}