#include "hip/hip_runtime.h"
//#ifndef __HIPCC__
//#define __HIPCC__
//#endif

#include <stdio.h>
#include <stdlib.h>
#include "globals_CUDA.h"
#include "declarations_CUDA.h"
//#include <hip/hip_runtime.h>
#include <>

__device__ void __forceinline__ MrqcofCurve23I0IA0(freq_context * __restrict__ CUDA_LCC, double * __restrict__ alpha, double * __restrict__ beta, int bid)
{
  int lpoints = 3;
  int mf1 = CUDA_mfit1;
  int l, jp, j, k, m, lnp2, Lpoints1 = lpoints + 1;
  double dy, sig2i, wt, ymod, wght, ltrial_chisq;
  __shared__ double dyda[BLOCKX4][N80];
  double * __restrict__ dydap = dyda[threadIdx.y];
  //__syncthreads();

  if(threadIdx.x == 0)
    {
      npg1[bid] += lpoints;
    }

  lnp2 = npg2[bid];
  ltrial_chisq = trial_chisqg[bid];

  int ma = CUDA_ma, lma = CUDA_lastma;
  int lastone = CUDA_lastone;
  int * __restrict__ iapp = CUDA_ia;
  double * __restrict__ dytemp = CUDA_LCC->dytemp, * __restrict__ ytemp = CUDA_LCC->ytemp;;
  
#pragma unroll 
  for(jp = 1; jp <= lpoints; jp++)
    {
      int ixx = jp + (threadIdx.x + 1) * Lpoints1; // ZZZ bad, strided read, BAD
      double * __restrict__ c = &(dytemp[ixx]);
      l = threadIdx.x;
#pragma unroll 2
      while(l < ma)
	{
	  dydap[l] = __ldca(c); // YYYY
	  l += CUDA_BLOCK_DIM;
	  c += CUDA_BLOCK_DIM * Lpoints1;
	}
      
      __syncwarp();
      
      lnp2++;
      double s = __ldg(&CUDA_sig[lnp2]);
      ymod = __ldca(&ytemp[jp]);
      sig2i = ___drcp_rn(s * s);
      wght = __ldg(&CUDA_Weight[lnp2]);
      dy = __ldg(&CUDA_brightness[lnp2]) - ymod;
      
      j = 0;
      double sig2iwght = sig2i * wght;
      
#pragma unroll 
      for(l = 2; l <= lastone; l++)
	{
	  j++;
	  wt = dydap[l-1] * sig2iwght;
	  
	  int xx = threadIdx.x + 1;
	  double * __restrict__ alp = &alpha[j * mf1 + xx - 1];
#pragma unroll 2
	  while(xx <= l)
	    {
	      //if(xx != 0)
	      double const * __restrict__ alp2 = alp;
	      __stwb(alp, *alp2 + wt * dydap[xx-1]);
	      xx  += CUDA_BLOCK_DIM;
	      alp += CUDA_BLOCK_DIM;
	    } /* m */
	  //__syncthreads();
	  if(threadIdx.x == 0)
	    {
	      beta[j] = beta[j] + dy * wt;
	    }
	  //__syncthreads();
	} /* l */
      
#pragma unroll 
      for(; l <= lma; l++)
	{
	  if(iapp[l])
	    {
	      j++;
	      wt = dydap[l-1] * sig2iwght;
	      
	      int xx = threadIdx.x + 1;
	      double * __restrict__ alph = &alpha[j * mf1 - 1];
#pragma unroll 2
	      while(xx <= lastone)
		{
		  //if(xx != 0)
		  __stwb(&alph[xx], alph[xx] + wt * dydap[xx-1]);
		  xx += CUDA_BLOCK_DIM;
		} /* m */
	      //__syncthreads();
	      if(threadIdx.x == 0)
		{
		  k = lastone - 1;
		  m = lastone + 1;
		  int * __restrict__ iap = iapp + m;
		  double * __restrict__ alp = alpha + j * mf1 + k;
#pragma unroll 4
		  for(; m <= l; m++)
		    {
		      if(*iap)
			{
			  alp++;
			  __stwb(alp, *alp + wt * dydap[m-1]);
			}
		      iap++;
		    } /* m */
		  beta[j] = beta[j] + dy * wt;
		}
	      //__syncthreads();
	    }
	} /* l */
      ltrial_chisq = ltrial_chisq + dy * dy * sig2iwght;
    } /* jp */

  if(threadIdx.x == 0)
    {
      npg2[bid] = lnp2;
      trial_chisqg[bid] = ltrial_chisq;
    }
}

__device__ void __forceinline__ MrqcofCurve23I0IA1(freq_context * __restrict__ CUDA_LCC, double * __restrict__ alpha, double * __restrict__ beta, int bid)
{
  int lpoints = 3;
  int mf1 = CUDA_mfit1;
  int l, jp, j, k, m, lnp2, Lpoints1 = lpoints + 1;
  double dy, sig2i, wt, ymod, wght, ltrial_chisq;
  __shared__ double dyda[N80];
  
  __syncwarp();

  if(threadIdx.x == 0)
    {
      npg1[bid] += lpoints;
    }

  lnp2 = npg2[bid];
  ltrial_chisq = trial_chisqg[bid];

  int ma = CUDA_ma, lma = CUDA_lastma;
  int lastone = CUDA_lastone;
  int * __restrict__ iapp = CUDA_ia;
  double * __restrict__ dytemp = CUDA_LCC->dytemp, * __restrict__ ytemp = CUDA_LCC->ytemp;
#pragma unroll 
  for(jp = 1; jp <= lpoints; jp++) 
    {
      lnp2++;
      double s = __ldg(&CUDA_sig[lnp2]);
      ymod = __ldca(&(ytemp[jp]));
      sig2i = ___drcp_rn(s * s); 
      wght = __ldg(&CUDA_Weight[lnp2]);
      dy = __ldg(&CUDA_brightness[lnp2]) - ymod;
      
      int ixx = jp + (threadIdx.x + 1) * Lpoints1; // ZZZ, bad, strided read, BAD!
      double * __restrict__ c = &(dytemp[ixx]); //  bad c
      l = threadIdx.x + 1;
#pragma unroll 4
      while(l <= ma - CUDA_BLOCK_DIM)
	{
	  double a, b;
	  a = __ldca(c);
	  c += CUDA_BLOCK_DIM * Lpoints1;
	  b = __ldca(c);
	  c += CUDA_BLOCK_DIM * Lpoints1;
	  dyda[l-1] = a;
	  dyda[l-1 + CUDA_BLOCK_DIM] = b;
	  l += 2*CUDA_BLOCK_DIM;
	}
#pragma unroll 1
      while(l <= ma)
	{
	  dyda[l-1] = __ldca(c);
	  l += CUDA_BLOCK_DIM;
	  c += CUDA_BLOCK_DIM * Lpoints1;
	}
      
      __syncwarp();
      
      j = 0;
      double sig2iwght = sig2i * wght;
      
#pragma unroll 4
      for(l = 1; l <= lastone; l++)
	{
	  j++;
	  wt = dyda[l-1] * sig2iwght;
	  int xx = threadIdx.x + 1;
#pragma unroll 2
	  while(xx <= l)
	    {
	      alpha[j * mf1 + xx] += wt * dyda[xx-1];
	      xx += CUDA_BLOCK_DIM;
	    } /* m */
	  //__syncthreads();
	  if(threadIdx.x == 0)
	    {
	      beta[j] = beta[j] + dy * wt;
	    }
	  //__syncthreads();
	} /* l */
      
#pragma unroll 4
      for(; l <= lma; l++)
	{
	  if(iapp[l])
	    {
	      j++;
	      wt = dyda[l-1] * sig2iwght;
	      int xx = threadIdx.x + 1;
#pragma unroll 2
	      while(xx <= lastone)
		{
		  //if(xx != 0)
		  alpha[j * mf1 + xx] += wt * dyda[xx-1];
		  xx += CUDA_BLOCK_DIM;
		} /* m */
	      //__syncthreads();
	      if(threadIdx.x == 0)
		{
		  k = lastone;
		  m = lastone + 1;
		  int * __restrict__ iap = iapp + m;
		  double * __restrict__ alp = alpha + j * mf1 + k;
#pragma unroll 4
		  for(; m <= l; m++)
		    {
		      if(*iap)
			{
			  alp++;
			  __stwb(alp, *alp + wt * dyda[m-1]);
			}
		      iap++;
		    } /* m */
		  beta[j] = beta[j] + dy * wt;
		}
	      //__syncthreads();
	    }
	} /* l */
      ltrial_chisq = ltrial_chisq + dy * dy * sig2iwght;
    } /* jp */

  if(threadIdx.x == 0)
    {
      npg2[bid] = lnp2;
      trial_chisqg[bid] = ltrial_chisq;
    }
}

__global__ void
__launch_bounds__(768) 
CudaCalculateIter1Mrqcof1Curve2I0IA0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I0IA0(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}


__global__ void
__launch_bounds__(768) 
CudaCalculateIter1Mrqcof1Curve2I0IA1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I0IA1(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}


__global__ void
__launch_bounds__(768) 
CudaCalculateIter1Mrqcof1Curve2I1IA0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I1IA0(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}

__global__ void
__launch_bounds__(768) 
CudaCalculateIter1Mrqcof1Curve2I1IA1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I1IA1(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}

__global__ 
void CudaCalculateIter1Mrqcof2CurveM12I0IA1(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  double *atryp = atry[bid]; //CUDA_LCC->atry;
  mrqcof_matrix(CUDA_LCC, atryp, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, atryp, 0, lpoints, bid);
  MrqcofCurve2I0IA1(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, lpoints, bid);
}

__global__ 
void CudaCalculateIter1Mrqcof2CurveM12I0IA0(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  double *atryp = atry[bid]; //CUDA_LCC->atry;
  mrqcof_matrix(CUDA_LCC, atryp, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, atryp, 0, lpoints, bid);
  MrqcofCurve2I0IA0(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, lpoints, bid);
}

__global__ void CudaCalculateIter1Mrqcof2Curve2I0IA0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  MrqcofCurve23I0IA0(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, bid);
}

__global__ void CudaCalculateIter1Mrqcof2Curve2I0IA1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  MrqcofCurve23I0IA1(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, bid);
}


// SLOW
__global__ void CudaCalculateIter1Mrqcof2Curve2I1IA0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  MrqcofCurve23I1IA0(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, bid);
}



__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof2Curve2I1IA1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  MrqcofCurve23I1IA1(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, bid);
}



__global__ 
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1CurveM12I0IA0(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  double *cg = CUDA_LCC->cg;
  mrqcof_matrix(CUDA_LCC, cg, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, cg, 0, lpoints, bid);
  MrqcofCurve2I0IA0(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, lpoints, bid);
}


__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1CurveM12I0IA1(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  double *cg = CUDA_LCC->cg;
  mrqcof_matrix(CUDA_LCC, cg, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, cg, 0, lpoints, bid);
  MrqcofCurve2I0IA1(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, lpoints, bid);
}



__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1CurveM12I1IA0(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  double *cg = CUDA_LCC->cg;
  mrqcof_matrix(CUDA_LCC, cg, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, cg, 1, lpoints, bid);
  MrqcofCurve2I1IA0(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, lpoints, bid);
}


__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1CurveM12I1IA1(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  double *cg = CUDA_LCC->cg;
  mrqcof_matrix(CUDA_LCC, cg, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, cg, 1, lpoints, bid);
  MrqcofCurve2I1IA1(CUDA_LCC, CUDA_LCC->alpha, CUDA_LCC->beta, lpoints, bid);
}


__global__ 
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1Curve1LastI0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  if(CUDA_LCC->ytemp == NULL) return;

  mrqcof_curve1_lastI0(CUDA_LCC, CUDA_LCC->cg, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}


__global__
__launch_bounds__(512) 
void CudaCalculateIter1Mrqcof1Curve1LastI1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;
  if(!__ldg(&isAlamda[bid])) return;

  mrqcof_curve1_lastI1(CUDA_LCC, CUDA_LCC->cg, CUDA_LCC->alpha, CUDA_LCC->beta, bid);
}

__global__ 
void CudaCalculateIter1Mrqcof2CurveM12I1IA1(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  double *atryp = atry[bid]; //CUDA_LCC->atry;
  mrqcof_matrix(CUDA_LCC, atryp, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, atryp, 1, lpoints, bid);
  MrqcofCurve2I1IA1(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, lpoints, bid);
}

__global__ 
__launch_bounds__(384) 
void CudaCalculateIter1Mrqcof2CurveM12I1IA0(const int lpoints)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  double *atryp = atry[bid]; //CUDA_LCC->atry;
  mrqcof_matrix(CUDA_LCC, atryp, lpoints, bid);
  mrqcof_curve1(CUDA_LCC, atryp, 1, lpoints, bid);
  MrqcofCurve2I1IA0(CUDA_LCC, CUDA_LCC->covar, CUDA_LCC->da, lpoints, bid);
}

__global__
__launch_bounds__(768) 
void CudaCalculateIter1Mrqcof2Curve1LastI0(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  mrqcof_curve1_lastI0(CUDA_LCC, atry[bid], CUDA_LCC->covar, CUDA_LCC->da, bid);
}

__global__
__launch_bounds__(1024) 
void CudaCalculateIter1Mrqcof2Curve1LastI1(void)
{
  int bid = blockIdx();
  auto CUDA_LCC = &CUDA_CC[bid];

  if(__ldg(&isInvalid[bid])) return;
  if(!__ldg(&isNiter[bid])) return;

  mrqcof_curve1_lastI1(CUDA_LCC, atry[bid], CUDA_LCC->covar, CUDA_LCC->da, bid);
}